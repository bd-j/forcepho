#include "hip/hip_runtime.h"
/* compute_gaussian_kernels.cu

This is the code to compute a Gaussian mixture likelihood and derivative
on the GPU.  Top-level code view:

Create chi^2 and d(chi2/dparam) accumulators in shared memory and zero them.

For each exposure:

    Create on-image Gaussians from on-sky parameters, put in shared memory
        ImageGaussian[NGalaxy*ImageGaussiansPerGalaxy]

    For one pixel per thread (taking BlockSize steps):

        Load Image Data
        Loop over all ImageGaussians:
            Evaluate Gaussians to create Residual image, save it

        Compute local_chi2 from residual image for this pixel
        Reduce local_chi2 over warp; atomic_add result to shared mem

        Loop over Active Galaxy:
            Loop over Gaussian in this Galaxy:
                Compute local_dchi_dp and accumulate
            Reduce local_dchi_dp over warp and atomic_add to shared dchi_dp for galaxy
            
When done with all exposures, copy the accumulators to the output buffer.
*/
// Shared memory is arranged in 32 banks of 4 byte stagger

//=============================================================================== 


#include "header.hh"
#include "patch.cu"
#include "proposal.cu"

// =====================  ImageGaussian class =============================

class ImageGaussian {
  public:
    // 6 Gaussian parameters
    float amp;
    float xcen; 
    float ycen;
    float fxx; 
    float fyy;
    float fxy; 
    
    // 15 Jacobian elements (Image -> Sky)
    float dA_dFlux;
    float dx_dAlpha;
    float dy_dAlpha;
    float dx_dDelta;
    float dy_dDelta;
    float dA_dQ;
    float dFxx_dQ;
    float dFyy_dQ;
    float dFxy_dQ;
    float dA_dPA;
    float dFxx_dPA;
    float dFyy_dPA;
    float dFxy_dPA;
    float dA_dSersic;
    float dA_drh;
};

// ======================  Code to Evaluate the Gaussians =========================

/// Compute the Model for one pixel from all galaxies; return the residual image
/// The one pixel is specified with xp, yp, data.
/// We have to enter a pointer to the whole list of ImageGaussians.

// TODO: n_gauss_total is a shared scalar in the calling function, but not here.
// Can we avoid the thread-based storage?  Max's advice is probably not.

__device__ PixFloat ComputeResidualImage(float xp, float yp, PixFloat data, ImageGaussian *g, int n_gauss_total)
{

    PixFloat residual = data;
    
    //loop over all image gaussians g for all galaxies. 
    for (int i = 0; i < n_gauss_total; i++, g++){
        // ImageGaussian *g = imageGauss+i;  // Now implicit in g++
        float dx = xp - g->xcen; 
        float dy = yp - g->ycen;

        float vx = g->fxx * dx + g->fxy * dy;
        float vy = g->fyy * dy + g->fxy * dx;
        float exparg = dx*vx + dy*vy;

        if (exparg>(float)MAX_EXP_ARG) continue;
        float Gp = expf(-0.5f * exparg);

        // Here are the second-order corrections to the pixel integral
        float H = 1.0f + (vx*vx + vy*vy - g->fxx - g->fyy) / 24.f; 
        float C = g->amp * Gp * H; //count in this pixel. 
        
        residual -= C; 
    }
    return residual;
}

// ====================  Code to Compute the Derivatives =====================

/// Compute the likelihood derivative for one pixel and one galaxy.
/// The one pixel is specified with xp, yp, and the residual*ierr^2.
/// We have to enter a pointer to the Gaussians for this one galaxy.
/// And we supply a pointer where we accumulate the derivatives.

// TODO: n_gal_gauss is a shared scalar in the calling function, but not here.
// Can we avoid the thread-based storage?  Max's advice is probably not.

__device__ void ComputeGaussianDerivative(float xp, float yp, float residual_ierr2, 
            ImageGaussian *g, float * dchi2_dp, int n_gal_gauss) 
{
    // Loop over all gaussians in this galaxy. 
    for (int gauss = 0; gauss<n_gal_gauss; gauss++, g++) {   
        // ImageGaussian *g = gaussian+gauss;  // Now implicit in g++
    
        float dx = xp - g->xcen; 
        float dy = yp - g->ycen; 
        float vx = g->fxx * dx + g->fxy * dy;
        float vy = g->fyy * dy + g->fxy * dx;
        
        float exparg = dx*vx + dy*vy; 
        if (exparg>(float)MAX_EXP_ARG) continue;
        
        float Gp = expf(-0.5f * exparg);
        float H = 1.0f + (vx*vx + vy*vy - g->fxx - g->fyy) *(1.0f/24.0f); 
    
        // Old code: this had divisions
        // float C = residual_ierr2 * g->amp * Gp * H;   
        // float dC_dA   = C / g->amp;
        // float c_h = C / H;

        float dC_dA = -2.f*residual_ierr2 * Gp;
        float c_h = dC_dA * g->amp;
        dC_dA *= H;
        float C   = c_h * H;
        float dC_dx   = C*vx;
        float dC_dy   = C*vy;
        float dC_dfx  = -0.5f*C*dx*dx;
        float dC_dfy  = -0.5f*C*dy*dy;
        float dC_dfxy = -1.0f*C*dx*dy;
    
        dC_dx    -= c_h * (g->fxx*vx + g->fxy*vy) * (1.0f/12.0f);
        dC_dy    -= c_h * (g->fyy*vy + g->fxy*vx) * (1.0f/12.0f);
        dC_dfx   -= c_h * (1.0f - 2.0f*dx*vx) * (1.0f/24.0f);
        dC_dfy   -= c_h * (1.0f - 2.0f*dy*vy) * (1.0f/24.0f);
        dC_dfxy  += c_h * (dy*vx + dx*vy) * (1.0f/12.0f);
             
        //Multiply by Jacobian and add to dchi2_dp    
        dchi2_dp[0] += g->dA_dFlux * dC_dA ; 
        dchi2_dp[1] += g->dx_dAlpha * dC_dx + g->dy_dAlpha * dC_dy;
        dchi2_dp[2] += g->dx_dDelta * dC_dx + g->dy_dDelta * dC_dy;
        dchi2_dp[3] += g->dA_dQ  * dC_dA + dC_dfx * g->dFxx_dQ + dC_dfxy * g->dFxy_dQ + dC_dfy * g->dFyy_dQ;
        dchi2_dp[4] += g->dA_dPA * dC_dA + dC_dfx * g->dFxx_dPA + dC_dfxy * g->dFxy_dPA + dC_dfy * g->dFyy_dPA;
        dchi2_dp[5] += g->dA_dSersic * dC_dA;
        dchi2_dp[6] += g->dA_drh * dC_dA;    
    }
}


// =================== Code to prepare the Gaussians =======================

/// This holds the information on a single source Gaussian, along with 
/// info needed to compute derivatives post-convolution.

typedef struct { 
    // 6 Gaussian parameters for sersic profile 
    float amp;
    float xcen;
    float ycen;
    float covar; //diagonal element of sersic profile covariance. covariance = covar * I. 
    matrix22 scovar_im; 
    
    //some distortions and astrometry specific to ??source??. 
    float flux;
    float G;
    float da_dn;
    float da_dr;
    matrix22 CW;
    matrix22 T;
    matrix22 dT_dq;
    matrix22 dT_dpa;
} PixGaussian; 

/// This function takes a source Gaussian and a PSF Gaussian and 
/// performs the convolution, creating the ImageGaussian that contains
/// the on-image gaussian plus the Jacobian to convert derivatives into
/// the on-sky parameters.

__device__ void  GetGaussianAndJacobian(PixGaussian & sersicgauss, PSFSourceGaussian & psfgauss, ImageGaussian & gauss){

    // sersicgauss.covar is the square of the radius; it's a scalar
    sersicgauss.scovar_im = sersicgauss.covar * AAt(sersicgauss.T); 
        
    matrix22 covar = sersicgauss.scovar_im + matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
    matrix22 f = covar.inv(); 
    float detF = f.det(); 
    
    gauss.fxx = f.v11; 
    gauss.fxy = f.v21; 
    gauss.fyy = f.v22; 
    
    gauss.xcen = sersicgauss.xcen + psfgauss.xcen; 
    gauss.ycen = sersicgauss.ycen + psfgauss.ycen; 
    
    float tmp = sersicgauss.G * psfgauss.amp * sqrtf(detF) * (1.0f/(2.0f*(float) M_PI));
    gauss.amp = tmp * sersicgauss.flux * sersicgauss.amp;
    // gauss.amp = sersicgauss.flux * sersicgauss.G * sersicgauss.amp * psfgauss.amp * sqrt(detF) * (1.0/(2.0*M_PI)) ;

    //now get derivatives of F
    matrix22 dSigma_dq  = sersicgauss.covar * symABt(sersicgauss.T, sersicgauss.dT_dq);
    // (sersicgauss.T * sersicgauss.dT_dq.T()+ sersicgauss.dT_dq*sersicgauss.T.T()); 
    matrix22 dSigma_dpa = sersicgauss.covar * symABt(sersicgauss.T, sersicgauss.dT_dpa);
    // (sersicgauss.T * sersicgauss.dT_dpa.T()+sersicgauss.dT_dpa*sersicgauss.T.T()); 
    
    matrix22 dF_dq      = -1.0f * ABA (f, dSigma_dq);  // F *  dSigma_dq * F
    matrix22 dF_dpa     = -1.0f * ABA (f, dSigma_dpa); // F * dSigma_dpa * F
    
    // Now get derivatives with respect to sky parameters
    // float ddetF_dq   = detF *  (covar * dF_dq).trace(); 
    // float ddetF_dpa  = detF * (covar * dF_dpa).trace(); 
    // gauss.dA_dQ      = gauss.amp /(2.0*detF) * ddetF_dq;  
    // gauss.dA_dPA     = gauss.amp /(2.0*detF) * ddetF_dpa;  
    // Old code: Why do we multiply and then divide by detF?

    gauss.dA_dQ      = 0.5f*gauss.amp * (covar * dF_dq).trace();
    gauss.dA_dPA     = 0.5f*gauss.amp * (covar * dF_dpa).trace();
    
    gauss.dA_dFlux      = tmp * sersicgauss.amp;
    gauss.dA_dSersic    = tmp * sersicgauss.flux * sersicgauss.da_dn;
    gauss.dA_drh        = tmp * sersicgauss.flux * sersicgauss.da_dr;

    // gauss.dA_dFlux   = gauss.amp / sersicgauss.flux; 
    // gauss.dA_dSersic = gauss.amp / sersicgauss.amp * sersicgauss.da_dn;
    // gauss.dA_drh     = gauss.amp / sersicgauss.amp * sersicgauss.da_dr;
    // Old code: Some opportunity in the above to avoid some divisions.
    
    gauss.dx_dAlpha = sersicgauss.CW.v11; 
    gauss.dy_dAlpha = sersicgauss.CW.v21; 
    
    gauss.dx_dDelta = sersicgauss.CW.v12;
    gauss.dy_dDelta = sersicgauss.CW.v22; 
    
    gauss.dFxx_dQ = dF_dq.v11;
    gauss.dFyy_dQ = dF_dq.v22;
    gauss.dFxy_dQ = dF_dq.v21; 

    gauss.dFxx_dPA = dF_dpa.v11;
    gauss.dFyy_dPA = dF_dpa.v22;
    gauss.dFxy_dPA = dF_dpa.v21; 
}

/// This function takes all of the sources for this exposure and 
/// convolves each with the PSFGaussians for all radii, producing a long
/// set of ImageGaussians that live in the shared memory.

__device__ void CreateImageGaussians(Patch * patch, Source * sources, int exposure, ImageGaussian * imageGauss) {
    
    // We're going to store some values common to the exposure in shared memory
    __shared__ int band, psfgauss_start, n_psf_per_source, n_gal_gauss; 
    __shared__ float G, crpix[2], crval[2]; 
    
    // Load the shared values
    if ( threadIdx.x == 0 ){
        band = blockIdx.x;   // This block is doing one band
        psfgauss_start = patch->psfgauss_start[exposure];
        G = patch->G[exposure]; 
    
        crpix[0] = patch->crpix[2*exposure]; crpix[1] = patch->crpix[2*exposure + 1];
        crval[0] = patch->crval[2*exposure]; crval[1] = patch->crval[2*exposure + 1];
    
        n_psf_per_source = patch->n_psf_per_source[band]; //constant per band. 
        n_gal_gauss = patch->n_sources * n_psf_per_source;
        // OPTION: Consider use of __constant__ variables
    }
    
    __syncthreads();
    
    // And now we're ready for the main loop.  
    // Each thread will work on one ImageGaussian, which means one PSF component
    // and one source radius for one galaxy/source.

    for (int tid = threadIdx.x; tid < n_gal_gauss; tid += blockDim.x) {
        // Unpack the source and gaussian.
        int g = tid / n_psf_per_source;       // Source number
        int p = tid - g * n_psf_per_source;   // Gaussian number
        
        Source *galaxy = sources+g;     
        PSFSourceGaussian *psfgauss = patch->psfgauss+psfgauss_start + p; 
        PixGaussian    sersicgauss;    // This is where we'll queue up the source info
    
        // Do the setup of the transformations        
        //Get the transformation matrix and other conversions
        matrix22 D, R, S; 
        
        int d_cw_start = 4 * (patch->n_sources * exposure + g); 
        D  = matrix22(patch->D+d_cw_start);        
        R.rot(galaxy->pa); 
        S.scale(galaxy->q); 
    
        //And its derivatives with respect to scene parameters
        matrix22 dS_dq, dR_dpa;
        dS_dq.scale_matrix_deriv(galaxy->q);
        dR_dpa.rotation_matrix_deriv(galaxy->pa);
                
        float smean[2]; 
        smean[0] = galaxy->ra  - crval[0];
        smean[1] = galaxy->dec - crval[1]; 
        sersicgauss.CW = matrix22(patch->CW+d_cw_start);
        Av(sersicgauss.CW, smean); //multiplies CW (2x2) by smean (2x1) and stores result in smean. 
        
        int s = psfgauss->sersic_radius_bin; 
        sersicgauss.xcen = smean[0] + crpix[0]; 
        sersicgauss.ycen = smean[1] + crpix[1]; 
        sersicgauss.covar = patch->rad2[s]; 
        sersicgauss.amp   = galaxy->mixture_amplitudes[s]; 
        sersicgauss.da_dn = galaxy->damplitude_dnsersic[s];
        sersicgauss.da_dr = galaxy->damplitude_drh[s] ; 
        sersicgauss.flux = galaxy->fluxes[band];         //pull the correct flux from the multiband array
        // G is the conversion of flux units to image counts
        sersicgauss.G = G; 
        // T is a unit circle, stretched by q, rotated by PA, and then distorted to the pixel scale
        sersicgauss.T = D * R * S; 
        // And now we have the derivatives of T wrt q and PA.
        sersicgauss.dT_dq  = D * R * dS_dq; 
        sersicgauss.dT_dpa = D * dR_dpa * S; 

        GetGaussianAndJacobian(sersicgauss, *psfgauss, imageGauss[tid]);
            // g * n_psf_per_source + p]);
    }
}
    
    

// ===================== Helper class for accumulating the results ========

class Accumulator {
  public:
    double chi2;
    float dchi2_dp[NPARAMS*MAXSOURCES]; 
        //OPTION: Figure out how to make this not compile time.

    __device__ Accumulator() {
    }
    __device__ ~Accumulator() { }

    __device__ void zero() {
        if (threadIdx.x==0) chi2 = 0.0;
        for (int j=threadIdx.x; j<NPARAMS*MAXSOURCES; j+=blockDim.x) dchi2_dp[j] = 0.0f;
        __syncthreads();
    }

    #define FULL_MASK 0xffffffff
    __device__ void warpReduceSum(float *answer, float input) {
        input += __shfl_down_sync(FULL_MASK, input, 16);
        input += __shfl_down_sync(FULL_MASK, input,  8);
        input += __shfl_down_sync(FULL_MASK, input,  4);
        input += __shfl_down_sync(FULL_MASK, input,  2);
        input += __shfl_down_sync(FULL_MASK, input,  1);
        // threadIdx.x % 32 == 0
        if ((threadIdx.x&31) == 0) atomicAdd(answer, input);
    }
    
    // Could put the Reduction code in here
    __device__ void SumChi2(float _chi2) {
        warpReduceSum(&chi2, _chi2);
    }
    __device__ void SumDChi2dp(float *_dchi2_dp, int gal) { 
        for (int j=0; j<NPARAMS; j++) 
            warpReduceSum(dchi2_dp+NPARAMS*gal+j, _dchi2_dp[j]); 
    }

    /// This copies this Accumulator into another memory buffer
    __device__ inline void store(float *pchi2, float *pdchi2_dp, int n_sources) {
        if (threadIdx.x==0) *pchi2 = chi2;
        for (int j=threadIdx.x; j<n_sources*NPARAMS; j+=blockDim.x)
            pdchi2_dp[j] = dchi2_dp[j];
        __syncthreads();
    }

    __device__ inline void addto(Accumulator &A, int n_sources) {
        if (threadIdx.x==0) chi2 += A.chi2;
        for (int j=threadIdx.x; j<n_sources*NPARAMS; j+=blockDim.x)
            dchi2_dp[j] += A.dchi2_dp[j];
        __syncthreads();
    }

    __device__ void coadd_and_sync(Accumulator *A, int nAcc, int n_sources) {
        for (int n=1; n<nAcc; n++) addto(A[n], n_sources);
            __syncthreads();
    }
};

// ================= Primary Proposal Kernel ========================


/// We are being handed pointers to a Patch structure, a Proposal structure,

/// a scalar chi2 response, and a vector dchi2_dp response.
/// The proposal is a pointer to Source[n_sources] sources.
/// The response is a pointer to [band][MaxSource] Responses.

#define THISBAND blockIdx.x
// Creating a more interpretable shorthand for this

extern "C" {
__global__ void EvaluateProposal(void *_patch, void *_proposal, 
                                 void *pchi2, void *pdchi2_dp) {
    // We will use a block of shared memory
    extern __shared__ char shared[];

    // Get the patch set up
    Patch *patch = (Patch *)_patch;  

    // The Proposal is a vector of Sources[n_sources]
    Source *sources = (Source *)_proposal;

    // Allocate the ImageGaussians for this band (same number for all exposures)
    __shared__ int n_gal_gauss;   // Number of image gaussians per galaxy
    __shared__ int band_N;   // The number of exposures in this band
    __shared__ int band_start;   // The starting exposures in this band
    __shared__ int n_sources;   // The number of sources
    __shared__ int n_gauss_total;   // Number of image gaussians for all sources
    __shared__ ImageGaussian *imageGauss; // [source][gauss]
    __shared__ Accumulator *accum;   // [NUMACCUMS]
    
    if (threadIdx.x==0) {
        int shared_counter = 0;

        n_gal_gauss = patch->n_psf_per_source[THISBAND];
        band_N = patch->band_N[THISBAND];
        band_start = patch->band_start[THISBAND];
        n_sources = patch->n_sources;
        n_gauss_total = n_sources*n_gal_gauss;
        accum = (Accumulator *) shared;
        shared_counter += NUMACCUMS*sizeof(Accumulator);
        imageGauss = (ImageGaussian *) (shared + shared_counter);
    }
    __syncthreads();   // Have to get this malloc done

    for (int j=0; j<NUMACCUMS; j++) accum[j].zero();
    float dchi2_dp[NPARAMS];   // This holds the derivatives for one galaxy

    // Now figure out which one this thread should use
    int threads_per_accum = ceilf(blockDim.x/warpSize/NUMACCUMS)*warpSize;
    int accumnum = threadIdx.x / threads_per_accum;  // We are accumulating each warp separately. 

    // Loop over Exposures
    for (int e = 0; e < band_N; e++) {
        int exposure = band_start + e;

        CreateImageGaussians(patch, sources, exposure, imageGauss);

        __syncthreads();
    
        for (int p = threadIdx.x ; p < patch->exposure_N[exposure]; p += blockDim.x) {
            int pix = patch->exposure_start[exposure] + p;

            // Get the data and compute the model for this one pixel
            float xp = patch->xpix[pix];
            float yp = patch->ypix[pix];
            PixFloat data = patch->data[pix];
            PixFloat ierr = patch->ierr[pix];
            PixFloat residual = ComputeResidualImage(xp, yp, data, imageGauss, n_gauss_total);

            // Did the CPU ask that we output the residual image?
            if(patch->residual != NULL)
                patch->residual[pix] = residual;

            // Compute chi2 and accumulate it
            residual *= ierr;   // Form residual/sigma, which is chi
            data *= ierr;
            // below computes (r^2 - d^2) / sigma^2 in an attempt to decrease the size of the
            // residual and avoid loss of significance 
            float chi2 = (residual - data);
            chi2 *= (residual + data);
            // chi2 -= 36.9;
            accum[accumnum].SumChi2((double) chi2);
            residual *= ierr;   // We want res*ierr^2 for the derivatives

            // Now we loop over Sources and compute the derivatives for each
            for (int gal = 0; gal < n_sources; gal++) {
                for (int j=0; j<NPARAMS; j++) dchi2_dp[j]=0.0f;
                ComputeGaussianDerivative(xp, yp, residual,  //1.
                        imageGauss+gal*n_gal_gauss, dchi2_dp, n_gal_gauss);  

                //if(gal == 0)
                //    patch->residual[pix] = dchi2_dp[1];

                accum[accumnum].SumDChi2dp(dchi2_dp, gal);

            }
        }

    __syncthreads();
    }

    // Now we're done with all exposures, but we need to sum the Accumulators
    // over all warps.
    accum[0].coadd_and_sync(accum, NUMACCUMS, n_sources);
    Response *r = (Response *)pdchi2_dp;
    accum[0].store((float *) pchi2 + THISBAND, (float *) &(r[THISBAND].dchi2_dparam), n_sources);
    return;
}

}  // extern "C"
