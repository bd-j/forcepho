#include "hip/hip_runtime.h"
/* 
NWarps = BlockSize/WarpSize(=32)

Create chi^2 and d(chi2/dparam) in shared memory and zero them.
    chi^2[NWarps]
    dchi_dp[NWarps][NActiveGalaxy]

For each exposure:

    Create on-image Gaussians from on-sky parameters, put in shared memory
    	ImageGaussian[NGalaxy*GaussianPerGalaxy]

    For one pixel per thread (taking BlockSize steps):

    	Load Image Data
		Loop over all ImageGaussians:
		    Evaluate Gaussians to create Residual image, save in register

		Compute local_chi2 from residual image for this pixel
		Reduce local_chi2 over warp; atomic_add result to shared mem

		Loop over Active Galaxy:
		    Loop over Gaussian in this Galaxy:
				Compute local_dchi_dp and accumulate
		    Reduce local_dchi_dp over warp and atomic_add to shared dchi_dp for galaxy
	    	
*/

/*
class Patch {
    // Exposure data[], ierr[], xpix[], ypix[], astrometry
    // List of FixedGalaxy
    // Number of SkyGalaxy
	int nActiveGals; 
	int nFixedGals;
	SkyGalaxy * FixedGals;
	//..list of fixed galaxies? 
	PixFloat * data;
	PixFloat * ierr;
	PixFloat * xpix; 
	PixFloat * ypix; 
	//..astrometry? 
	PSFGaussian * psfgauss;
	int * n_psf_gauss;   // Indexed by exposure
	int * start_psf_gauss;   // Indexed by exposure
};

class SkyGalaxy {
    // On-sky galaxy parameters
	// flux: total flux
	// ra: right ascension (degrees)
	// dec: declination (degrees)
	// q, pa: axis ratio squared and position angle
	// n: sersic index
	// r: half-light radius (arcsec)
	float flux; 
	float ra; 
	float dec;
	float q; 
	float pa; 
	float n;
	float r; 
};
class Proposal {
	SkyGalaxy * ActiveGals;     // List of SkyGalaxy, input for this HMC likelihood
};

typedef struct{
	int nGauss; 
	ImageGaussian * gaussians; //List of ImageGaussians
    // TODO: May not need this to be explicit
} Galaxy;
*/  
//=================== ABOVE THIS LINE IS DEPRECATED ============


#include "header.hh"
#include "patch.cu"
#include "proposal.cu"

//NAM do we want this class, or should we make the convolve a method of PSFSourceGaussian?
typedef struct { //NAM TODO write a constructor? 
    // 6 Gaussian parameters
	float amp;
	float xcen;
	float ycen;
	float Cxx;
	float Cyy;
	float Cxy;
	
	float flux; 
	float G; 
	matrix22 CW; 
	
	matrix22 T; 
	matrix22 dT_dq;
	matrix22 dT_dpa;
	
	matrix22 scovar_im; 
	matrix22 scovar; 
	
	float da_dn;
	float da_dr; 
	
	// TODO: Consider whether a dummy float will help with shared memory bank constraints
} PixGaussian;

//NAM if all galaxies active, don't need to separate these... 
typedef struct {
    // 6 Gaussian parameters
	float amp;
	float xcen; 
	float ycen;
	float fxx; 
	float fyy;
	float fxy; 
	// TODO: Consider whether a dummy float will help with shared memory bank constraints
	
    // 15 Jacobian elements (Image -> Sky)
    float dA_dFlux;
    float dx_dAlpha;
    float dy_dAlpha;
    float dx_dDelta;
    float dy_dDelta;
    float dA_dQ;
    float dFxx_dQ;
    float dFyy_dQ;
    float dFxy_dQ;
    float dA_dPA;
    float dFxx_dPA;
    float dFyy_dPA;
    float dFxy_dPA;
    float dA_dSersic;
    float dA_drh;
} ImageGaussian;

// typedef struct {
//     // 15 Jacobian elements (Image -> Sky)
//     float dA_dFlux;
//     float dx_dAlpha;
//     float dy_dAlpha;
//     float dx_dDelta;
//     float dy_dDelta;
//     float dA_dQ;
//     float dFxx_dQ;
//     float dFyy_dQ;
//     float dFxy_dQ;
//     float dA_dPA;
//     float dFxx_dPA;
//     float dFyy_dPA;
//     float dFxy_dPA;
//     float dA_dSersic;
//     float dA_drh;
// } ImageGaussianJacobian;


__device__ PixFloat ComputeResidualImage(float xp, float yp, PixFloat data, Patch patch, Galaxy galaxy); //NAM do we need patch, galaxy? 
{
	PixFloat residual = data;
	
	// TODO: Need to loop over Active and Fixed galaxies
	//loop over all image gaussians g. 
	for (int i = 0; i < galaxy.nGauss; i ++){ //NAM TODO nGauss may be derived from Patch class properties. 
		ImageGaussian g = galaxy.Gaussians[i]
		float dx = xp - g.xcen; 
		float dy = yp - g.ycen; 
		float vx = g.fxx * dx + g.fxy * dy;
		float vy = g.fyy * dy + g.fxy * dx;
		float exparg = dx*vx+dy*vy;
		if (exparg>MAX_EXP_ARG) continue;
		float Gp = exp(-0.5 * exparg);

		// Here are the second-order corrections to the pixel integral
		float H = 1.0 + (vx*vx + vy*vy - g.fxx - g.fyy) / 24.0; 
		float C = g.amp * Gp * H; //count in this pixel. 
		
		residual -= C; 
	}
	return residual;
}

__device__ void ComputeGaussianDerivative(pix, xp, yp, residual, gal, gauss, float * dchi2_dp) //NAM why are we passing in residual? it's been accumulated over ImageGaussians earlier... we need to repeat work here to get isolated Image Gaussian
{
	float dx = xp - gauss.xcen; 
	float dy = yp - gauss.ycen; 
	float vx = gauss.fxx * dx + gauss.fxy * dy;
	float vy = gauss.fyy * dy + gauss.fxy * dx;
	float Gp = exp(-0.5 * (dx*vx + dy*vy));
	float H = 1.0; 
	float root_det = 1.0; 			
	
	H = 1.0 + (vx*vx + vy*vy - gauss.fxx - gauss.fyy) / 24.0; 
	float C = gauss.amp * Gp * H * root_det; //count in this pixel. 
	
    float dC_dA = C / gauss.amp;
    float dC_dx = C*vx;
    float dC_dy = C*vy;
    float dC_dfx = -0.5*C*dx*dx;
    float dC_dfy = -0.5*C*dy*dy;
    float dC_dfxy = -1.0*C*dx*dy;
	
    float c_h = C / H;
    dC_dx -= c_h * (g.fxx*vx + g.fxy*vy) / 12.0;
    dC_dy -= c_h * (g.fyy*vy + g.fxy*vx) / 12.0;
    dC_dfx -= c_h * (1.0 - 2.0*dx*vx) / 24.0;
    dC_dfy -= c_h * (1.0 - 2.0*dy*vy) / 24.0;
    dC_dfxy += c_h * (dy*vx + dx*vy) / 12.0;
	
	
	 // gradients += np.matmul(g.derivs, dI_dphi)
//      gradients[0][:] = dC_dA[:]
//      gradients[1][:] = dC_dx[:]
//      gradients[2][:] = dC_dy[:]
//      gradients[3][:] = dC_dfx[:]
//      gradients[4][:] = dC_dfy[:]
//      gradients[5][:] = dC_dfxy[:]
			 
			 
	//NAM TODO this is wrong. 

    dchi2_dpim[0] += residual * dC_dA; 
    dchi2_dpim[1] += residual * dC_dx;
    dchi2_dpim[2] += residual * dC_dy;
    dchi2_dpim[3] += residual * dC_dfx;
    dchi2_dpim[4] += residual * dC_dfy;
    dchi2_dpim[5] += residual * dC_dfxy;

    // TODO: Multiply by Jacobian and add to dchi2_dp
}


#define NACTIVE MAXSOURCES   // Hack for now

void warpReduceSum(float *answer, float input) {
    input += __shfl_down(input, 16);
    input += __shfl_down(input,  8);
    input += __shfl_down(input,  4);
    input += __shfl_down(input,  2);
    input += __shfl_down(input,  1);
    if (threadIdx.x&31==0) *answer = input;
}

class Accumulator {
  public:
    float chi2;
    float dchi2_dp[NPARAM*NACTIVE]; //TODO: Need to figure out how to make this not compile time.

    Accumulator() {
	chi2 = 0.0;
	for (int j=0; j<NPARAM*NACTIVE; j++) dchi2_dp[j] = 0.0;
    }
    ~Accumulator() { }
    
    // Could put the Reduction code in here
    void SumChi2(float _chi2) { warpReduceSum(&chi2, _chi2); }
    void SumDChi2dp(float *_dchi2_dp, int gal) { 
	for (int j=0; j<NPARAM; j++) 
	    warpReduceSum(dchi2_dp+j+NPARAM*gal, _dchi2_dp[j]); 
    }

    /// This copies this Accumulator into another memory buffer
    inline void store(float *pchi2, float *pdchi2_dp, int nActive) {
        if (threadIdx.x==0) *pchi2 = chi2;
	for (int j=threadIdx.x; j<nActive*NPARAM; j+=BlockDim.x)
	    pdchi2_dp[j] = dchi2_dp[j];
    }

    inline void addto(Accumulator &A) {
        if (threadIdx.x==0) chi2 += A.chi2;
	for (int j=threadIdx.x; j<nActive*NPARAM; j+=BlockDim.x)
	    dchi2_dp[j] += A.dchi2_dp[j];
    }

    void coadd_and_sync(Accumulator *A, int nAcc) {
        for (int n=1; n<nAcc; n++) addto(A[n]);
	__syncthreads();
    }
};


__device__ void  GetGaussianAndJacobian(PixGaussian sersicgauss, PSFSourceGaussian psfgauss, ImageGaussian & gauss){
	sersicgauss.scovar_im = T * sersicgauss.scovar * T.T();
	
	matrix22 covar = sersicgauss.scovar_im + matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
	matrix22 f = covar.inv(); 
	
	gauss.fxx = f.v11; 
	gauss.fxy = f.v21; 
	gauss.fyy = f.v22; 
	
	gauss.xcen = sersicgauss.xcen + psfgauss.xcen; 
	gauss.ycen = sersicgauss.ycen + psfgauss.ycen; 
	
	gauss.amp = sersicgauss.flux * sersicgauss.amp * psfgauss.amp * pow(f.det(), 0.5) / (2.0 * math.pi) ;
	
	matrix22 Sigma = sersicgauss.scovar_im + matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
	matrix22 F = Sigma.inv(); 
	float detF = F.det(); 
	float K = sersicgauss.flux * sersicgauss.G * sersicgauss.amp * psfgauss.amp * pow(detF, 0.5) / (2.0 * math.pi); 
	
	//now get derivatives 
	//of F
	matrix22 dSigma_dq  = sersicgauss.T * sersicgauss.scovar * sersicgauss.dT_dq.T() + sersicgauss.dT_dq * sersicgauss.scovar * sersicgauss.T.T(); 
	matrix22 dSigma_dpa = sersicgauss.T * sersicgauss.scovar * sersicgauss.dT_dpa.T() + sersicgauss.dT_dpa * sersicgauss.scovar * sersicgauss.T.T(); 
	matrix22 dF_dq      = -F * dSigma_dq * F; 
	matrix22 dF_dpa     = -F * dSigma_dpa * F; 
	float ddetF_dq   = detF *  (Sigma * dF_dq).trace(); 
	float ddetF_dpa  = detF * (Sigma * dF_dpa).trace(); 
	
	//of Amplitude
    gauss.dA_dQ = K / (2.0 * detF) * ddetF_dq;  
    gauss.dA_dpA = K / (2.0 * detF) * ddetF_dpa;  
    gauss.dA_dFlux = K / sersicgauss.flux; 
    gauss.dA_dSersic = K / sersicgauss.amp * sersicgauss.da_dn;
    gauss.dA_drh = K / sersicgauss.amp * sersicgauss.da_dr;
	
	gauss.dx_dAlpha = CW.v11; 
	gauss.dy_dAlpha = CW.v21; 
	
	gauss.dx_dDelta = CW.v12;
	gauss.dy_dDelta = CW.v22; 
	
	gauss.dFxx_dQ = dF_dq.v11;
	gauss.dFyy_dQ = dF_dq.v22;
	gauss.dFxy_dQ = dF_dq.v21; 

	gauss.dFxx_dPA = dF_dpa.v11;
	gauss.dFyy_dPA = dF_dpa.v22;
	gauss.dFxy_dPA = dF_dpa.v21; 
}



// __device__ void ConstructImageGaussian(matrix22 scovar, matrix22 pcovar,
// 												matrix22 smean, float samp, matrix22 pmean, float pamp, float flux, ImageGaussian &gauss){
// 	matrix22 covar = scovar + pcovar;
// 	matrix22 f = covar.inv();
//
// 	gauss.fxx = f.v11;
// 	gauss.fxy = f.v21;
// 	gauss.fyy = f.v22;
//
// 	gauss.xcen = smean.v11 + pmean.v11; //NAM careful!
// 	gauss.ycen = smean.v22 + pmean.v22;
//
// 	gauss.amp = flux * samp * pamp * pow(f.det(), 0.5) / (2.0 * math.pi) ;
// }
//
// __device__ void ConstructImageJacobian(matrix22 scovar, matrix22 pcovar,
// 												float samp, float pamp, float flux, float G, matrix22 T, matrix22 dT_dq, matrix22 dT_dpa, float da_dn, float da_dr, matrix22 CW, ImageGaussianJacobian &jacobian){
//
// 	//convolve the s-th Source component with the p-th PSF component.
// 	matrix22 scovar_im = T * scovar * T.T();
// 	matrix22 Sigma = scovar_im + pcovar;
// 	matrix22 F = Sigma.inv();
// 	float detF = F.det();
// 	float K = flux * G * samp * pamp * pow(detF, 0.5) / (2.0 * math.pi);
//
// 	//now get derivatives
// 	//of F
// 	matrix22 dSigma_dq  = T * scovar * dT_dq.T() + dT_dq * scovar * T.T();
// 	matrix22 dSigma_dpa = T * scovar * dT_dpa.T() + dT_dpa * scovar * T.T();
// 	matrix22 dF_dq      = -F * dSigma_dq * F;
// 	matrix22 dF_dpa     = -F * dSigma_dpa * F;
// 	float ddetF_dq   = detF *  (Sigma * dF_dq).trace();
// 	float ddetF_dpa  = detF * (Sigma * dF_dpa).trace();
//
// 	//of Amplitude
//     jacobian.dA_dQ = K / (2.0 * detF) * ddetF_dq;
//     jacobian.dA_dpA = K / (2.0 * detF) * ddetF_dpa;
//     jacobian.dA_dFlux = K / flux;
//     jacobian.dA_dSersic = K / samp * da_dn;
//     jacobian.dA_drh = K / samp * da_dr;
//
// 	jacobian.dx_dAlpha = CW.v11;
// 	jacobian.dy_dAlpha = CW.v21;
//
// 	jacobian.dx_dDelta = CW.v12;
// 	jacobian.dy_dDelta = CW.v22;
//
// 	jacobian.dFxx_dQ = dF_dq.v11;
// 	jacobian.dFyy_dQ = dF_dq.v22;
// 	jacobian.dFxy_dQ = dF_dq.v21;
//
// 	jacobian.dFxx_dPA = dF_dpa.v11;
// 	jacobian.dFyy_dPA = dF_dpa.v22;
// 	jacobian.dFxy_dPA = dF_dpa.v21;
// }

__device__ void CreateImageGaussians(Patch * patch, Source * sources, int exposure) {
	
	int tid = threadIdx.x; 
    int band = blockIdx.x;   // This block is doing one band
	
	int psfgauss_start = patch->psfgauss_start[exposure];
	
	__shared__ float G = patch->G[exposure]; 
	__shared__ float crpix[2], crval[2]; 
	
	crpix[0] = patch->crpix[exposure][0];  crpix[1] = patch->crpix[exposure][1];  
	crval[0] = patch->crval[exposure][0];  crval[1] = patch->crval[exposure][1]; 
	
	int n_psf_per_source = patch->n_psf_per_source[band]; //constant per band. 
	//int n_radii = patch->n_radii;
	
	while (tid < patch->n_sources * n_psf_per_source){
        int g = tid / n_psf_per_source;
		int p = tid - g * n_psf_per_source;
		
		Source galaxy = sources[g]; 	
		PSFSourceGaussian psfgauss = patch->psfgauss[psfgauss_start + p]; 
		PixGaussian	sersicgauss; 
		
		sersicgauss.G = G; 
		
		int s = psfgauss.sersic_radius_bin; 
			
	    // Do the setup of the transformations		
		//Get the transformation matrix and other conversions
		matrix22 D, R, S; 
		
		int d_cw_start = 4 * patch->n_sources * exposure + 4 * g; 
		D  = matrix22(patch->D[d_cw_start ], patch->D[d_cw_start  + 1], patch->D[d_cw_start  + 2], patch->D[d_cw_start  + 3]); // NAM yuck! 
		sersicgauss.CW = matrix22(patch->CW[d_cw_start], patch->CW[d_cw_start + 1], patch->CW[d_cw_start + 2], patch->CW[d_cw_start + 3]);
		
		R.rot(galaxy.pa); 
		S.scale(galaxy.q); 
		sersicgauss.T = D * R * S; 
	
		//And its derivatives with respect to scene parameters
		matrix22 dS_dq, dR_dpa;
		dS_dq.scale_matrix_deriv(galaxy.q);
		dR_dpa.rotation_matrix_deriv(galaxy.pa);
		sersicgauss.dT_dq  = D * R * dS_dq; 
		sersicgauss.dT_dpa = D * dR_dpa * S; 	
	
		//NAM  might benefit from a vector class. this is gross. 
		float smean[2]; 
		smean[0] = galaxy.ra  - patch->crval[exposure][0];
		smean[1] = galaxy.dec - patch->crval[exposure][1]; 
	    matrix22::Av(CW, *smean);
		
		sersicgauss.xcen = smean[0] + patch->crpix[exposure][0]; 
		sersicgauss.ycen = smean[1] + patch->crpix[exposure][1]; 
		
		sersicgauss.Cxx = patch->rad2[s]; 
		sersicgauss.Cyy = patch->rad2[s]; 
		sersicgauss.Cxy = 0.0; 
		
		sersicgauss.scovar = matrix22(sersicgauss.Cxx, sersicgauss.Cxy, sersicgauss.Cxy, sersicgauss.Cyy );
		
		sersicgauss.amp = galaxy.mixture_amplitudes[s]; 
		
		sersicgauss.da_dn = galaxy.damplitude_dnsersic[s];
		sersicgauss.da_dr = galaxy.damplitude_drh[s] ; 

		//pull the correct flux from the multiband array
		sersicgaus.flux = proposal.fluxes[blockId.x]; 
		//float pamp = psfgauss.amp; 

		//get PSF component means and covariances in the pixel space
		//NAM TODO assumes units are already in pixel space. 
		//matrix22 pcovar = matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
	    //matrix22 pmean  = matrix22(psfgauss.xcen, psfgauss.cen); 

    	//ConstructImageGaussian(T * scovar T.T(), pcovar, smean, samp, pmean, pamp, flux, imageGauss[gal*n_gal_gauss+s*n_psf_gauss+p]);

    	GetGaussianAndJacobian(sersicgauss, psfgauss, imageGauss[gal * n_psf_per_source + p]);
				
    		//ConstructImageJacobian(scovar, pcovar, samp, pamp, flux, G, T, dT_dq, dT_dpa, da_dn, da_dr, CW, imageJacob[gal*n_gal_gauss+s*n_psf_gauss+p]);
		
		
		tid += blockDim.x; 
	}
	
	

	
	
	
	//     for (int gal=0; gal<nActiveGals+nFixedGals; gal++) {
	// 	Patch patch = ;//NAM TODO
	//
	//     // Do the setup of the transformations
	// 	//Get the transformation matrix and other conversions
	// 	matrix22 D = matrix22(patch.scale[0], patch.scale[1]); //diagonal 2x2 matrix.
	// 	matrix22 R = rot(galaxy.pa);
	// 	matrix22 S = scale(galaxy.q);
	// 	matrix22 T = D * R * S;
	// 	matrix22 CW = matrix22(patch.dpix_dsky[0], patch.dpix_dsky[1]);
	// 	float G = patch.photocounts;
	//
	// 	//And its derivatives with respect to scene parameters
	// 	matrix22 dS_dq = scale_matrix_deriv(galaxy.q);
	// 	matrix22 dR_dpa = rotation_matrix_deriv(galaxy.pa);
	// 	matrix22 dT_dq = D * R * dS_dq;
	// 	matrix22 dT_dpa = D * dR_dpa * S;
	//
	//
	//         for (int s=0; s<patch->nSersicGauss; s++) {
	// 		//get source spline and derivatives
	// 	    smean = patch.sky_to_pix([source.ra, source.dec]) //NAM TODO	//these don't have to be matrix22s. just two numbers...
	//
	// 		matrix22 scovar = matrix22(galaxy.covariances[s], galaxy.covariances[s]) ; //diagonal elements of this gaussian's covariance matrix for sersic index s.
	// 		float samp = galaxy.amplitudes[s];
	// 		float da_dn = galaxy.damplitude_dsersic[s];
	// 		float da_dr = galaxy.damplitude_drh[s] ;
	//
	// 		//pull the correct flux from the multiband array
	// 		float flux = patch.flux[blockId.x]; //NAM TODO is this right?
	//
	//     	for (int p=0; p<n_psf_gauss; p++) {
	// 			float pamp = patch.psf.amplitudes[p];
	//
	// 			//get PSF component means and covariances in the pixel space
	// 			if (patch.psf.units[p] == 'arcsec'){
	// 				matrix22 pcovar = D * patch.psf.covariances[p] * D.T();
	// 				matrix22 pmean = D * patch.psf.means[p];  //these don't have to be matrix22s. just two numbers...
	// 			}
	// 			else if (patch.psf.units == 'pixels'){
	// 				matrix22 pcovar = patch.psf.covariances[p];
	// 		        matrix22 pmean = stamp.psf.means[p]; //these don't have to be matrix22s. just two numbers...
	// 			}
	//
	// 	    	ConstructImageGaussian(s,p,gal, T * scovar T.T(), pcovar, smean, samp, pmean, pamp, flux, imageGauss[gal*n_gal_gauss+s*n_psf_gauss+p]);
	//
	// 			if (gal<nActiveGals) {
	// 	    		ConstructImageJacobian(s,p,gal, scovar, pcovar, samp, pamp, flux, G, T, dT_dq, dT_dpa, da_dn, da_dr, CW, imageJacob[gal*n_gal_gauss+s*n_psf_gauss+p]);
	// 			}
	//     	}
	//     	}
	// }
}


// ================= Primary Proposal Kernel ========================

// Shared memory is arranged in 32 banks of 4 byte stagger

/// We are being handed pointers to a Patch structure, a Proposal structure,

/// a scalar chi2 response, and a vector dchi2_dp response.
/// The proposal is a pointer to Source[n_active] sources.
/// The response is a pointer to [band][MaxSource] responses.

__global__ void EvaluateProposal(void *_patch, void *_proposal, 
                                 void *pchi2, void *pdchi2_dp) {
    // Get the patch set up
    Patch *patch = (Patch *)_patch;  

    // The Proposal is a vector of Sources[n_active]
    Source *sources = (Source *)_proposal;

    // TODO: THIS IS BROKEN.
    // Need to define a shared pointer and then have one thread
    // call malloc to allocate this shared memory.
    // CreateAndZeroAccumulators();
    __shared__ Accumulator accum[blockDim.x/WARPSIZE]();
    int warp = threadIdx.x / WARPSIZE;  // We are accumulating each warp separately. 
	
    int band = blockIdx.x;   // This block is doing one band

    // Loop over Exposures
    for (e = 0; e < patch->band_N[band]; e++) {
        int exposure = patch->band_start[band] + e;
		int start_psf_gauss = patch->psfgauss_start[exposure];

        // TODO: THIS IS BROKEN.
        // Need to define a shared pointer and then have one thread
        // call malloc to allocate this shared memory.
		int n_gal_gauss = patch->n_psf_per_source[band];
		__shared__ ImageGaussians imageGauss[n_gal_gauss * patch->n_sources];
		__shared__ ImageGaussiansJacobians imageJacob[n_gal_gauss * patch->n_sources];

        CreateImageGaussians(patch, sources, exposure);

		__syncthreads();
	
		for (p = threadIdx.x ; p < patch->exposure_N[exposure]; p += blockDim.x) {
		    int pix = patch->exposure_start[exposure] + p;

		    float xp = patch->xpix[pix];
		    float yp = patch->ypix[pix];
		    PixFloat data = patch->data[pix];
		    PixFloat ierr = patch->ierr[pix];
		    PixFloat residual = ComputeResidualImage(xp, yp, data); 
		    // This loads data and ierr, then subtracts the active
		    // and fixed Gaussians to make the residual

		    float chi2 = residual*ierr;
		    chi2 *= chi2;
		    accum[warp].SumChi2(chi2);
		    /// ReduceWarp_Add(chi2, accum[warp].chi2));
	    
		    // Now we loop over Active Galaxies and compute the derivatives
		    for (gal = 0; gal < patch.n_sources; gal++) {
		    		float dchi2_dp[NPARAM];
				for (int j=0; j<NPARAM; j++) dchi2_dp[j]=0.0;
				for (gauss = 0; ) {
				    ComputeGaussianDerivative(pix, residual, gal, gauss, dchi2_dp); 
				}
			
				accum[warp].SumDChi2dp(dchi2_dp, gal);

				///ReduceWarp_Add(dchi2_dp, accum[warp].dchi2_dp);
		    }
		}
	__syncthreads();
    }

    // Now we're done with all exposures, but we need to sum the Accumulators
    // over all warps.
    accum[0].coadd_and_sync(accum, blockDim.x/WARPSIZE);
    Response *r = (Response *)pdchi2_dp;
    accum[0].store((float *)pchi2, &(pdchi2_dp[blockIdx.x].dchi2_dparam), patch->n_sources);
    return;
}
