#include "hip/hip_runtime.h"
/* compute_gaussian_kernels.cu

This is the code to compute a Gaussian mixture likelihood and derivative
on the GPU.  Top-level code view:

Create chi^2 and d(chi2/dparam) accumulators in shared memory and zero them.

For each exposure:

    Create on-image Gaussians from on-sky parameters, put in shared memory
    	ImageGaussian[NGalaxy*GaussianPerGalaxy]

    For one pixel per thread (taking BlockSize steps):

    	Load Image Data
		Loop over all ImageGaussians:
		    Evaluate Gaussians to create Residual image, save it

		Compute local_chi2 from residual image for this pixel
		Reduce local_chi2 over warp; atomic_add result to shared mem

		Loop over Active Galaxy:
		    Loop over Gaussian in this Galaxy:
				Compute local_dchi_dp and accumulate
		    Reduce local_dchi_dp over warp and atomic_add to shared dchi_dp for galaxy
	    	
When done with all exposures, copy the accumulators to the output buffer.
*/
//=================== ABOVE THIS LINE IS DEPRECATED ============


#include "header.hh"
#include "patch.cu"
#include "proposal.cu"

//NAM do we want this class, or should we make the convolve a method of PSFSourceGaussian?
typedef struct { 
    // 6 Gaussian parameters for sersic profile 
	float amp;
	float xcen;
	float ycen;
	float covar; //diagonal element of sersic profile covariance. covariance = covar * I. 
	matrix22 scovar_im; 
	
	//some distortions and astrometry specific to ??source??. 
	float flux; 
	float G; 
	float da_dn;
	float da_dr;
	matrix22 CW; 
	matrix22 T; 
	matrix22 dT_dq;
	matrix22 dT_dpa;
} PixGaussian; 


class ImageGaussian {
  public:
    // 6 Gaussian parameters
	float amp;
	float xcen; 
	float ycen;
	float fxx; 
	float fyy;
	float fxy; 
	
    // 15 Jacobian elements (Image -> Sky)
    float dA_dFlux;
    float dx_dAlpha;
    float dy_dAlpha;
    float dx_dDelta;
    float dy_dDelta;
    float dA_dQ;
    float dFxx_dQ;
    float dFyy_dQ;
    float dFxy_dQ;
    float dA_dPA;
    float dFxx_dPA;
    float dFyy_dPA;
    float dFxy_dPA;
    float dA_dSersic;
    float dA_drh;
};


__device__ PixFloat ComputeResidualImage(float xp, float yp, PixFloat data, ImageGaussian * imageGauss, int n_gauss_total); 
{
	PixFloat residual = data;
	
	//loop over all image gaussians g for all galaxies. 
	for (int i = 0; i < n_gauss_total; i ++){
		ImageGaussian g = imageGauss[i]
		float dx = xp - g.xcen; 
		float dy = yp - g.ycen; 
		float vx = g.fxx * dx + g.fxy * dy;
		float vy = g.fyy * dy + g.fxy * dx;
		float exparg = dx*vx+dy*vy;
		if (exparg>MAX_EXP_ARG) continue;
		float Gp = exp(-0.5 * exparg);

		// Here are the second-order corrections to the pixel integral
		float H = 1.0 + (vx*vx + vy*vy - g.fxx - g.fyy) / 24.0; 
		float C = g.amp * Gp * H; //count in this pixel. 
		
		residual -= C; 
	}
	return residual;
}

__device__ void ComputeGaussianDerivative(float xp, float yp, float residual_ierr2, 
            ImageGaussian *gaussian, float * dchi2_dp) //pass in pointer to first gaussian for this galaxy. 
{
	for (int gauss = 0; gauss<n_gal_gauss; gauss++) {   //loop ovver all gaussians in this galaxy. 
		ImageGaussian g = gaussian[gauss];
	
		float dx = xp - g->xcen; 
		float dy = yp - g->ycen; 
		float vx = g->fxx * dx + g->fxy * dy;
		float vy = g->fyy * dy + g->fxy * dx;
		float Gp = exp(-0.5 * (dx*vx + dy*vy));
	
		float H = 1.0 + (vx*vx + vy*vy - g->fxx - g->fyy) / 24.0; 
		float C = residual_ierr2 * g->amp * Gp * H;   
	
	    float dC_dA   = C / g->amp;
	    float dC_dx   = C*vx;
	    float dC_dy   = C*vy;
	    float dC_dfx  = -0.5*C*dx*dx;
	    float dC_dfy  = -0.5*C*dy*dy;
	    float dC_dfxy = -1.0*C*dx*dy;
	
	    float c_h = C / H;
	    dC_dx    -= c_h * (g->fxx*vx + g->fxy*vy) / 12.0;
	    dC_dy    -= c_h * (g->fyy*vy + g->fxy*vx) / 12.0;
	    dC_dfx   -= c_h * (1.0 - 2.0*dx*vx) / 24.0;
	    dC_dfy   -= c_h * (1.0 - 2.0*dy*vy) / 24.0;
	    dC_dfxy  += c_h * (dy*vx + dx*vy) / 12.0;
			 
	    //Multiply by Jacobian and add to dchi2_dp	
		dchi2_dp[0] += g.dA_dFlux * dC_dA ; 
		dchi2_dp[1] += g.dx_dAlpha * dC_dx + g.dy_dAlpha * dC_dy;
		dchi2_dp[2] += g.dx_dDelta * dC_dx + g.dy_dDelta * dC_dy;
		dchi2_dp[3] += g.dA_dQ  * dC_dA + dC_dfx * g.dFxx_dQ + dC_dfxy * g.dFxy_dQ + dC_dfy * g.dFyy_dQ;
		dchi2_dp[4] += g.dA_dPA * dC_dA + dC_dfx * g.dFxx_dPA + dC_dfxy * dFxy_dPA + dC_dfy * dFyy_dPA;
		dchi2_dp[5] += g.dA_dSersic * dC_dA;
		dchi2_dp[6] += g.dA_drh * dC_dA;	
	}
}


class Accumulator {
  public:
    float chi2;
    float dchi2_dp[NPARAMS*MAXSOURCES]; //TODO: Need to figure out how to make this not compile time.
	//NAM TODO NPARAM=7 is baked into some assumptions above... changing it will break things. 

    Accumulator() {
        chi2 = 0.0;
        for (int j=0; j<NPARAMS*MAXSOURCES; j++) dchi2_dp[j] = 0.0;
    }
    ~Accumulator() { }

    void warpReduceSum(float *answer, float input) {
        input += __shfl_down(input, 16);
        input += __shfl_down(input,  8);
        input += __shfl_down(input,  4);
        input += __shfl_down(input,  2);
        input += __shfl_down(input,  1);
        if (threadIdx.x&31==0) atomicAdd_block(answer, input);
    }
    
    // Could put the Reduction code in here
    void SumChi2(float _chi2) { warpReduceSum(&chi2, _chi2); }
    void SumDChi2dp(float *_dchi2_dp, int gal) { 
        for (int j=0; j<NPARAMS; j++) 
            warpReduceSum(dchi2_dp+NPARAMS*gal+j, _dchi2_dp[j]); 
    }

    /// This copies this Accumulator into another memory buffer
    inline void store(float *pchi2, float *pdchi2_dp, int nActive) {
        if (threadIdx.x==0) *pchi2 = chi2;
        for (int j=threadIdx.x; j<nActive*NPARAMS; j+=BlockDim.x)
            pdchi2_dp[j] = dchi2_dp[j];
    }

    inline void addto(Accumulator &A) {
        if (threadIdx.x==0) chi2 += A.chi2;
        for (int j=threadIdx.x; j<nActive*NPARAMS; j+=BlockDim.x)
            dchi2_dp[j] += A.dchi2_dp[j];
    }

    void coadd_and_sync(Accumulator *A, int nAcc) {
        for (int n=1; n<nAcc; n++) addto(A[n]);
        __syncthreads();
    }
};


__device__ void  GetGaussianAndJacobian(PixGaussian sersicgauss, PSFSourceGaussian psfgauss, ImageGaussian & gauss){
	
	sersicgauss.scovar_im = sersicgauss.covar * T.AAt(); 
		
	matrix22 covar = sersicgauss.scovar_im + matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
	matrix22 f = covar.inv(); 
	float detF = f.det(); 
	
	gauss.fxx = f.v11; 
	gauss.fxy = f.v21; 
	gauss.fyy = f.v22; 
	
	gauss.xcen = sersicgauss.xcen + psfgauss.xcen; 
	gauss.ycen = sersicgauss.ycen + psfgauss.ycen; 
	
	gauss.amp = sersicgauss.flux * sersicgauss.G * sersicgauss.amp * psfgauss.amp * sqrt(detF) / (2.0 * math.pi) ;

	//now get derivatives 
	//of F
	matrix22 dSigma_dq  = sersicgauss.covar * (sersicgauss.T * sersicgauss.dT_dq.T()  + sersicgauss.dT_dq  * sersicgauss.T.T() ) ; 
	matrix22 dSigma_dpa = sersicgauss.covar * (sersicgauss.T * sersicgauss.dT_dpa.T() + sersicgauss.dT_dpa * sersicgauss.T.T() ) ; 
	
	matrix22 dF_dq      = -matrix22::ABA (F, dSigma_dq);  // F *  dSigma_dq * F
	matrix22 dF_dpa     = -matrix22::ABA (F, dSigma_dpa); // F * dSigma_dpa * F
	
	float ddetF_dq   = detF *  (Sigma * dF_dq).trace(); 
	float ddetF_dpa  = detF * (Sigma * dF_dpa).trace(); 
	
	//of Amplitude
    gauss.dA_dQ      = gauss.amp / (2.0 * detF) * ddetF_dq;  
    gauss.dA_dpA     = gauss.amp / (2.0 * detF) * ddetF_dpa;  
    gauss.dA_dFlux   = gauss.amp / sersicgauss.flux; 
    gauss.dA_dSersic = gauss.amp / sersicgauss.amp * sersicgauss.da_dn;
    gauss.dA_drh     = gauss.amp / sersicgauss.amp * sersicgauss.da_dr;
	
	gauss.dx_dAlpha = sersicgauss.CW.v11; 
	gauss.dy_dAlpha = sersicgauss.CW.v21; 
	
	gauss.dx_dDelta = sersicgauss.CW.v12;
	gauss.dy_dDelta = sersicgauss.CW.v22; 
	
	gauss.dFxx_dQ = dF_dq.v11;
	gauss.dFyy_dQ = dF_dq.v22;
	gauss.dFxy_dQ = dF_dq.v21; 

	gauss.dFxx_dPA = dF_dpa.v11;
	gauss.dFyy_dPA = dF_dpa.v22;
	gauss.dFxy_dPA = dF_dpa.v21; 
}


__device__ void CreateImageGaussians(Patch * patch, Source * sources, int exposure, ImageGaussian * imageGauss) {
	
	__shared__ int band, psfgauss_start, n_psf_per_source, n_radii, n_gal_gauss; 
	__shared__ float G, crpix[2], crval[2]; 
	
	
	if ( threadIdx.x == 0 ){
	    band = blockIdx.x;   // This block is doing one band
		psfgauss_start = patch->psfgauss_start[exposure];
		G = patch->G[exposure]; 
	
		crpix[0] = patch->crpix[exposure][0];  crpix[1] = patch->crpix[exposure][1];  
		crval[0] = patch->crval[exposure][0];  crval[1] = patch->crval[exposure][1]; 
	
		n_psf_per_source = patch->n_psf_per_source[band]; //constant per band. 
		n_radii = patch->n_radii;
	    n_gal_gauss = patch->n_sources * n_psf_per_source;
	    // TODO: Consider use of __constant__ variables
	}
	
	__syncthreads();
	

	for (int tid = threadIdx.x; tid < n_gal_gauss; tid += blockDim.x) {
        int g = tid / n_psf_per_source;       // Source number
		int p = tid - g * n_psf_per_source;   // Gaussian number
		
		Source *galaxy = sources+g; 	
		PSFSourceGaussian *psfgauss = patch->psfgauss+psfgauss_start + p; 
		PixGaussian	sersicgauss; 
	
	    // Do the setup of the transformations		
		//Get the transformation matrix and other conversions
		matrix22 D, R, S; 
		
		int d_cw_start = 4 * (patch->n_sources * exposure + g); 
		D  = matrix22(patch->D+d_cw_start);		
		R.rot(galaxy.pa); 
		S.scale(galaxy.q); 
	
		//And its derivatives with respect to scene parameters
		matrix22 dS_dq, dR_dpa;
		dS_dq.scale_matrix_deriv(galaxy.q);
		dR_dpa.rotation_matrix_deriv(galaxy.pa);
			
	
		//NAM  might benefit from a vector class. this is gross. 
		float smean[2]; 
		smean[0] = galaxy.ra  - crval[0];
		smean[1] = galaxy.dec - crval[1]; 
		sersicgauss.CW = matrix22(patch->CW+d_cw_start);
	    matrix22::Av(sersicgauss.CW, *smean); //multiplies CW (2x2) by smean (2x1) and stores result in smean. 
		
		int s = psfgauss->sersic_radius_bin; 
		sersicgauss.xcen = smean[0] + crpix[0]; 
		sersicgauss.ycen = smean[1] + crpix[1]; 
		sersicgauss.covar = patch->rad2[s]; 
		sersicgauss.amp   = galaxy.mixture_amplitudes[s]; 
		sersicgauss.da_dn = galaxy.damplitude_dnsersic[s];
		sersicgauss.da_dr = galaxy.damplitude_drh[s] ; 
		sersicgauss.flux = galaxy->fluxes[band]; 		//pull the correct flux from the multiband array
		sersicgauss.G = G; 
		sersicgauss.T = D * R * S; 
		sersicgauss.dT_dq  = D * R * dS_dq; 
		sersicgauss.dT_dpa = D * dR_dpa * S; 

    	GetGaussianAndJacobian(sersicgauss, psfgauss, imageGauss[gal * n_psf_per_source + p]);
	}
}
	
	


// ================= Primary Proposal Kernel ========================

// Shared memory is arranged in 32 banks of 4 byte stagger

/// We are being handed pointers to a Patch structure, a Proposal structure,

/// a scalar chi2 response, and a vector dchi2_dp response.
/// The proposal is a pointer to Source[n_active] sources.
/// The response is a pointer to [band][MaxSource] Responses.

__global__ void EvaluateProposal(void *_patch, void *_proposal, 
                                 void *pchi2, void *pdchi2_dp) {
    // Get the patch set up
    Patch *patch = (Patch *)_patch;  

    // The Proposal is a vector of Sources[n_active]
    Source *sources = (Source *)_proposal;

    // Create And Zero Accumulators
    __shared__ Accumulator accum[NUMACCUMS]();

    // Now figure out which one this thread should use
    // TODO: 32 is the warp size; perhaps use a built-in name
    int threads_per_accum = ceilf(blockDim.x/32/NUMACCUMS)*32;    
    int accumnum = threadIdx.x / threads_per_accum;  // We are accumulating each warp separately. 
	
    int thisband = blockIdx.x;   // This block is doing one band
    // TODO: Would this be better as a #define?  I.e., perhaps the blockIdx is faster/lighter

    // Allocate the ImageGaussians for this band (same number for all exposures)
    int n_gal_gauss = patch->n_psf_per_source[thisband];
    __shared__ ImageGaussian *imageGauss; // [source][gauss]
    if (threadIDx.x==0) 
        imageGauss = (ImageGaussian *)malloc(
                sizeof(ImageGaussian)*n_gal_gauss * patch->n_sources);
        // The claim is that this malloc returns shared memory because the 
        // target pointer is shared.
    __syncthreads();   // Have to get this malloc done

    // Loop over Exposures
    for (int e = 0; e < patch->band_N[thisband]; e++) {
        int exposure = patch->band_start[thisband] + e;
		int start_psf_gauss = patch->psfgauss_start[exposure];

        CreateImageGaussians(patch, sources, exposure, imageGauss);

		__syncthreads();
	
		for (int p = threadIdx.x ; p < patch->exposure_N[exposure]; p += blockDim.x) {
		    int pix = patch->exposure_start[exposure] + p;

		    float xp = patch->xpix[pix];
		    float yp = patch->ypix[pix];
		    PixFloat data = patch->data[pix];
		    PixFloat ierr = patch->ierr[pix];
		    PixFloat residual = ComputeResidualImage(xp, yp, data, imageGauss, n_gal_gauss * patch->n_sources); 
            patch->residual[pix] = residual;

            residual *= ierr;   // Form residual/sigma, which is chi
		    float chi2 = residual*residual;
		    accum[accumnum].SumChi2(chi2);
            residual *= ierr;   // We want res*ierr^2 for the derivatives
	    
		    // Now we loop over Active Galaxies and compute the derivatives
		    for (int gal = 0; gal < patch.n_sources; gal++) {
                float dchi2_dp[NPARAM]; //NAM i think this only needs to be declared once per a thread's lifetime, so long as it's zeroed below. 
				for (int j=0; j<NPARAM; j++) dchi2_dp[j]=0.0;
				ComputeGaussianDerivative(xp, yp, residual, imageGauss+gal*n_gal_gauss, dchi2_dp);  //loop over all gaussians
				accum[accumnum].SumDChi2dp(dchi2_dp, gal);
		    }
		}
	__syncthreads();
    }

    // Now we're done with all exposures, but we need to sum the Accumulators
    // over all warps.
    accum[0].coadd_and_sync(accum, NUMACCUMS);
    Response *r = (Response *)pdchi2_dp;
    accum[0].store((float *)pchi2, &(pdchi2_dp[thisband].dchi2_dparam), patch->n_sources);
    return;
}
