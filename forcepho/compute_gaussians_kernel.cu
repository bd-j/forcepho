#include "hip/hip_runtime.h"
/* 
NWarps = BlockSize/WarpSize(=32)

Create chi^2 and d(chi2/dparam) in shared memory and zero them.
    chi^2[NWarps]
    dchi_dp[NWarps][NActiveGalaxy]

For each exposure:

    Create on-image Gaussians from on-sky parameters, put in shared memory
        ImageGaussian[NGalaxy*GaussianPerGalaxy]

    For one pixel per thread (taking BlockSize steps):

        Load Image Data
        Loop over all ImageGaussians:
            Evaluate Gaussians to create Residual image, save in register

        Compute local_chi2 from residual image for this pixel
        Reduce local_chi2 over warp; atomic_add result to shared mem

        Loop over Active Galaxy:
            Loop over Gaussian in this Galaxy:
                Compute local_dchi_dp and accumulate
            Reduce local_dchi_dp over warp and atomic_add to shared dchi_dp for galaxy
            
*/

/*
class Patch {
    // Exposure data[], ierr[], xpix[], ypix[], astrometry
    // List of FixedGalaxy
    // Number of SkyGalaxy
    int nActiveGals; 
    int nFixedGals;
    SkyGalaxy * FixedGals;
    //..list of fixed galaxies? 
    PixFloat * data;
    PixFloat * ierr;
    PixFloat * xpix; 
    PixFloat * ypix; 
    //..astrometry? 
    PSFGaussian * psfgauss;
    int * nPSFGauss;   // Indexed by exposure
    int * startPSFGauss;   // Indexed by exposure
};

class SkyGalaxy {
    // On-sky galaxy parameters
    // flux: total flux
    // ra: right ascension (degrees)
    // dec: declination (degrees)
    // q, pa: axis ratio squared and position angle
    // n: sersic index
    // r: half-light radius (arcsec)
    float flux; 
    float ra; 
    float dec;
    float q; 
    float pa; 
    float n;
    float r; 
};
class Proposal {
    SkyGalaxy * ActiveGals;     // List of SkyGalaxy, input for this HMC likelihood
};

typedef struct{
    int nGauss; 
    ImageGaussian * gaussians; //List of ImageGaussians
    // TODO: May not need this to be explicit
} Galaxy;
*/  
//=================== ABOVE THIS LINE IS DEPRECATED ============

typedef float PixFloat;
#define NPARAM 7    // Number of Parameters per Galaxy, in one band 
#define MAXACTIVE 30    // Max number of active Galaxies in a patch

typedef struct {
    // 6 Gaussian parameters
    float amp;
    float xcen; 
    float ycen;
    float Cxx; 
    float Cyy;
    float Cxy; 
    // TODO: Consider whether a dummy float will help with shared memory bank constraints
} PixGaussian;

typedef struct {
    // 6 Gaussian parameters
    float amp;
    float xcen; 
    float ycen;
    float fxx; 
    float fyy;
    float fxy; 
    // TODO: Consider whether a dummy float will help with shared memory bank constraints
} ImageGaussian;

typedef struct {
    // 15 Jacobian elements (Image -> Sky)
    float dA_dFlux;
    float dx_dAlpha;
    float dy_dAlpha;
    float dx_dDelta;
    float dy_dDelta;
    float dA_dQ;
    float dFxx_dQ;
    float dFyy_dQ;
    float dFxy_dQ;
    float dA_dPA;
    float dFxx_dPA;
    float dFyy_dPA;
    float dFxy_dPA;
    float dA_dSersic;
    float dA_drh;
} ImageGaussianJacobian;


#define MAX_EXP_ARG 36.0

__device__ PixFloat ComputeResidualImage(float xp, float yp, PixFloat data, Patch patch, Galaxy galaxy); //NAM do we need patch, galaxy? 
{
    PixFloat residual = data;
    
    // TODO: Need to loop over Active and Fixed galaxies
    //loop over all image gaussians g. 
    for (int i = 0; i < galaxy.nGauss; i ++){ //NAM TODO nGauss may be derived from Patch class properties. 
        ImageGaussian g = galaxy.Gaussians[i]
        float dx = xp - g.xcen; 
        float dy = yp - g.ycen; 
        float vx = g.fxx * dx + g.fxy * dy;
        float vy = g.fyy * dy + g.fxy * dx;
        float exparg = dx*vx+dy*vy;
        if (exparg>MAX_EXP_ARG) continue;
        float Gp = exp(-0.5 * exparg);

        // Here are the second-order corrections to the pixel integral
        float H = 1.0 + (vx*vx + vy*vy - g.fxx - g.fyy) / 24.0; 
        float C = g.amp * Gp * H; //count in this pixel. 
        
        residual -= C; 
    }
    return residual;
}

__device__ void ComputeGaussianDerivative(pix, xp, yp, residual, gal, gauss, float * dchi2_dp) //NAM why are we passing in residual? it's been accumulated over ImageGaussians earlier... we need to repeat work here to get isolated Image Gaussian
{
    float dx = xp - gauss.xcen; 
    float dy = yp - gauss.ycen; 
    float vx = gauss.fxx * dx + gauss.fxy * dy;
    float vy = gauss.fyy * dy + gauss.fxy * dx;
    float Gp = exp(-0.5 * (dx*vx + dy*vy));
    float H = 1.0; 
    float root_det = 1.0;             
    
    H = 1.0 + (vx*vx + vy*vy - gauss.fxx - gauss.fyy) / 24.0; 
    float C = gauss.amp * Gp * H * root_det; //count in this pixel. 
    
    float dC_dA = C / gauss.amp;
    float dC_dx = C*vx;
    float dC_dy = C*vy;
    float dC_dfx = -0.5*C*dx*dx;
    float dC_dfy = -0.5*C*dy*dy;
    float dC_dfxy = -1.0*C*dx*dy;
    
    float c_h = C / H;
    dC_dx -= c_h * (g.fxx*vx + g.fxy*vy) / 12.0;
    dC_dy -= c_h * (g.fyy*vy + g.fxy*vx) / 12.0;
    dC_dfx -= c_h * (1.0 - 2.0*dx*vx) / 24.0;
    dC_dfy -= c_h * (1.0 - 2.0*dy*vy) / 24.0;
    dC_dfxy += c_h * (dy*vx + dx*vy) / 12.0;
    
    
    gradients += np.matmul(g.derivs, dI_dphi)
    gradients[0][:] = dC_dA[:]
    gradients[1][:] = dC_dx[:]
    gradients[2][:] = dC_dy[:]
    gradients[3][:] = dC_dfx[:]
    gradients[4][:] = dC_dfy[:]
    gradients[5][:] = dC_dfxy[:]
             
             
             

    dchi2_dpim[0] += residual * dC_dA; //NAM TODO ??  is this right? 
    dchi2_dpim[1] += residual * dC_dx;
    dchi2_dpim[2] += residual * dC_dy;
    dchi2_dpim[3] += residual * dC_dfx;
    dchi2_dpim[4] += residual * dC_dfy;
    dchi2_dpim[5] += residual * dC_dfxy;

    // TODO: Multiply by Jacobian and add to dchi2_dp
}


#define NACTIVE MAXACTIVE   // Hack for now

void warpReduceSum(float *answer, float input) {
    input += __shfl_down(input, 16);
    input += __shfl_down(input,  8);
    input += __shfl_down(input,  4);
    input += __shfl_down(input,  2);
    input += __shfl_down(input,  1);
    if (threadIdx.x&31==0) *answer = input;
}

class Accumulator {
  public:
    float chi2;
    float dchi2_dp[NPARAM*NACTIVE]; //TODO: Need to figure out how to make this not compile time.

    Accumulator() {
        chi2 = 0.0;
        for (int j=0; j<NPARAM*NACTIVE; j++) dchi2_dp[j] = 0.0;
    }
    ~Accumulator() { }
    
    // Could put the Reduction code in here
    void SumChi2(float _chi2) { warpReduceSum(&chi2, _chi2); }
    void SumDChi2dp(float *_dchi2_dp, int gal) { 
    for (int j=0; j<NPARAM; j++) 
        warpReduceSum(dchi2_dp+j+NPARAM*gal, _dchi2_dp[j]); 
    }

    /// This copies this Accumulator into another memory buffer
    inline void store(float *pchi2, float *pdchi2_dp, int nActive) {
        if (threadIdx.x==0) *pchi2 = chi2;
        for (int j=threadIdx.x; j<nActive*NPARAM; j+=BlockDim.x)
            pdchi2_dp[j] = dchi2_dp[j];
    }

    inline void addto(Accumulator &A) {
        if (threadIdx.x==0) chi2 += A.chi2;
        for (int j=threadIdx.x; j<nActive*NPARAM; j+=BlockDim.x)
            dchi2_dp[j] += A.dchi2_dp[j];
    }

    void coadd_and_sync(Accumulator *A, int nAcc) {
        for (int n=1; n<nAcc; n++) addto(A[n]);
        __syncthreads();
    }
};





__device__ void ConstructImageGaussian(int s, int p, Galaxy gal, matrix22 scovar, matrix22 pcovar, 
                                                matrix22 smean, float samp, matrix22 pmean, float pamp, float flux, ImageGaussian &gauss){
    matrix22 covar = scovar + pcovar; 
    matrix22 f = covar.inv(); 
    
    gauss.fxx = f.v11; 
    gauss.fxy = f.v21; 
    gauss.fyy = f.v22; 
    
    gauss.xcen = smean.v11 + pmean.v11; //NAM careful! 
    gauss.ycen = smean.v22 + pmean.v22; 
    
    gauss.amp = flux * samp * pamp * pow(f.det(), 0.5) / (2.0 * math.pi) ;
}

__device__ void ConstructImageJacobian(int s, int p, Galaxy gal, matrix22 scovar, matrix22 pcovar, 
                                                float samp, float pamp, float flux, float G, matrix22 T, matrix22 dT_dq, matrix22 dT_dpa, float da_dn, float da_dr, matrix22 CW, ImageGaussianJacobian &jacobian){

    //convolve the s-th Source component with the p-th PSF component.
    matrix22 scovar_im = T * scovar * T.T();
    matrix22 Sigma = scovar_im + pcovar; 
    matrix22 F = Sigma.inv(); 
    float detF = F.det(); 
    float K = flux * G * samp * pamp * pow(detF, 0.5) / (2.0 * math.pi); 
    
    //now get derivatives 
    //of F
    matrix22 dSigma_dq  = T * scovar * dT_dq.T() + dT_dq * scovar * T.T(); 
    matrix22 dSigma_dpa = T * scovar * dT_dpa.T() + dT_dpa * scovar * T.T(); 
    matrix22 dF_dq      = -F * dSigma_dq * F; 
    matrix22 dF_dpa     = -F * dSigma_dpa * F; 
    float ddetF_dq   = detF *  (Sigma * dF_dq).trace(); 
    float ddetF_dpa  = detF * (Sigma * dF_dpa).trace(); 
    
    //of Amplitude
    jacobian.dA_dQ = K / (2.0 * detF) * ddetF_dq;  
    jacobian.dA_dpA = K / (2.0 * detF) * ddetF_dpa;  
    jacobian.dA_dFlux = K / flux; 
    jacobian.dA_dSersic = K / am * da_dn;
    jacobian.dA_drh = K / am * da_dr;
    
    jacobian.dx_dAlpha = CW.v11; 
    jacobian.dy_dAlpha = CW.v21; 
    
    jacobian.dx_dDelta = CW.v12;
    jacobian.dy_dDelta = CW.v22; 
    
    jacobian.dFxx_dQ = dF_dq.v11;
    jacobian.dFyy_dQ = dF_dq.v22;
    jacobian.dFxy_dQ = dF_dq.v21; 

    jacobian.dFxx_dPA = dF_dpa.v11;
    jacobian.dFyy_dPA = dF_dpa.v22;
    jacobian.dFxy_dPA = dF_dpa.v21; 
}

__device__ void CreateImageGaussians() {
    
    int tid = threadIdx.x; 
    
    int totGals = nActiveGals+nFixedGals; 
    int tmp = (tid - p * patch->nSersicGauss * totGals); 

    int p = tid / patch->nSersicGauss * totGals; 
    int s = tmp/totGals; 
    int g = tmp%totGals; 
    
    
    
    
    
    
    
    for (int gal=0; gal<nActiveGals+nFixedGals; gal++) {
        Patch patch = ;//NAM TODO
        
        // Do the setup of the transformations        
        //Get the transformation matrix and other conversions
        // TODO: Probably not implementing the sky distortions in full
        matrix22 D = matrix22(patch.scale[0], patch.scale[1]); //diagonal 2x2 matrix. 
        matrix22 R = rot(galaxy.pa);   // TODO: fix calling syntax
        matrix22 S = scale(galaxy.q); 
        matrix22 T = D * R * S; 
        matrix22 CW = matrix22(patch.dpix_dsky[0], patch.dpix_dsky[1]);
        float G = patch.photocounts; 
        
        //And its derivatives with respect to scene parameters
        matrix22 dS_dq = scale_matrix_deriv(galaxy.q);
        matrix22 dR_dpa = rotation_matrix_deriv(galaxy.pa);
        matrix22 dT_dq = D * R * dS_dq; 
        matrix22 dT_dpa = D * dR_dpa * S;     
        
        
        for (int s=0; s<patch->nSersicGauss; s++) {
            //get source spline and derivatives
            smean = patch.sky_to_pix([source.ra, source.dec]) //NAM TODO    //these don't have to be matrix22s. just two numbers... 
                    
            matrix22 scovar = matrix22(galaxy.covariances[s], galaxy.covariances[s]) ; //diagonal elements of this gaussian's covariance matrix for sersic index s. 
            float samp = galaxy.amplitudes[s]; 
            float da_dn = galaxy.damplitude_dsersic[s];
            float da_dr = galaxy.damplitude_drh[s] ; 
            
            //pull the correct flux from the multiband array
            float flux = patch.flux[blockId.x]; //NAM TODO is this right? 
            
            for (int p=0; p<nPSFGauss; p++) {
                float pamp = patch.psf.amplitudes[p]; 
                
                //get PSF component means and covariances in the pixel space
                if (patch.psf.units[p] == 'arcsec'){
                    matrix22 pcovar = D * patch.psf.covariances[p] * D.T();
                    matrix22 pmean = D * patch.psf.means[p];  //these don't have to be matrix22s. just two numbers... 
                }
                else if (patch.psf.units == 'pixels'){
                    matrix22 pcovar = patch.psf.covariances[p]; 
                    matrix22 pmean = stamp.psf.means[p]; //these don't have to be matrix22s. just two numbers... 
                }
                
                ConstructImageGaussian(s,p,gal, T * scovar T.T(), pcovar, smean, samp, pmean, pamp, flux, imageGauss[gal*nGalGauss+s*nPSFGauss+p]);
                
                if (gal<nActiveGals) {
                    ConstructImageJacobian(s,p,gal, scovar, pcovar, samp, pamp, flux, G, T, dT_dq, dT_dpa, da_dn, da_dr, CW, imageJacob[gal*nGalGauss+s*nPSFGauss+p]);
                }
            }
        }
    }
}


// ================= Primary Proposal Kernel ========================

// Shared memory is arranged in 32 banks of 4 byte stagger
#define WARPSIZE 32

/// We are being handed pointers to a Patch structure, a Proposal structure,
/// a scalar chi2 response, and a vector dchi2_dp response
__global__ void EvaluateProposal(void *_patch, void *_proposal, 
                                 void *pchi2, void *pdchi2_dp) {
    Patch *patch = (Patch *)_patch;  // We should be given this pointer

    // The Proposal is a vector of Sources[n_active]
    Source *sources = (Source *)_proposal;

    int band = blockIdx.x;   // This block is doing one band
    int warp = threadIdx.x / WARPSIZE;  // We are accumulating in warps.

    // Create and Zero Accumulators();
    __shared__ Accumulator accum[blockDim.x/WARPSIZE]();

    // Loop over Exposures
    for (e = 0; e < patch->NumExposures[band]; e++) {
        int exposure = patch->StartExposures[band] + e;
        int nPSFGauss = patch->nPSFGauss[exposure];
        int startPSFGauss = patch->startPSFGauss[exposure];
        int nGalGauss = nPSFGauss*patch->nSersicGauss;

        __shared__ ImageGaussians imageGauss[nGalGauss*(nActiveGals+nFixedGals)];
            // Convention is Active first, then Fixed.
        __shared__ ImageGaussiansJacobians imageJacob[nGalGauss*(nActiveGals)];
            // We only need the Active galaxies
            CreateImageGaussians(patch, exposure);

        __syncthreads();

        for (p = threadIdx.x ; p < patch->NumPixels[exposure]; p += blockDim.x) {
            int pix = patch->StartPixels[exposure] + p;

            // Get the data for this pixel
            float xp = patch.xpix[pix];
            float yp = patch.ypix[pix];
            PixFloat data = patch.data[pix];
            PixFloat ierr = patch.ierr[pix];

            // Subtracts the active and fixed Gaussians to make the residual
            PixFloat residual = ComputeResidualImage(xp, yp, data); 

            // Compute chi2 and accumulate it
            float chi2 = residual*ierr;
            chi2 *= chi2;
            accum[warp].SumChi2(chi2);
        
            // Now we loop over Active Galaxies and compute the derivatives
            for (gal = 0; gal < patch.nActiveGals; gal++) {
            float dchi2_dp[NPARAM];
            for (int j=0; j<NPARAM; j++) dchi2_dp[j]=0.0;
            for (gauss = 0; ) {    // TODO: Finish this
                ComputeGaussianDerivative(pix, residual, gal, gauss, dchi2_dp); 
            }
            accum[warp].SumDChi2dp(dchi2_dp, gal);
            }
        } // Done looping over pixels in this exposure
        __syncthreads();
    }

    // Now we're done with all exposures, but we need to sum the Accumulators
    // over all warps.
    accum[0].coadd_and_sync(accum, blockDim.x/WARPSIZE);
    accum[0].store((float *)pchi2, (float *)pdchi2_dp, nActiveGals);
    // TODO: This needs to offset the given vectors according to the Band!
    return;
}
