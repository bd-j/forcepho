#include "hip/hip_runtime.h"
/* 
NWarps = BlockSize/WarpSize(=32)

Create chi^2 and d(chi2/dparam) in shared memory and zero them.
    chi^2[NWarps]
    dchi_dp[NWarps][NActiveGalaxy]

For each exposure:

    Create on-image Gaussians from on-sky parameters, put in shared memory
    	ImageGaussian[NGalaxy*GaussianPerGalaxy]

    For one pixel per thread (taking BlockSize steps):

    	Load Image Data
		Loop over all ImageGaussians:
		    Evaluate Gaussians to create Residual image, save in register

		Compute local_chi2 from residual image for this pixel
		Reduce local_chi2 over warp; atomic_add result to shared mem

		Loop over Active Galaxy:
		    Loop over Gaussian in this Galaxy:
				Compute local_dchi_dp and accumulate
		    Reduce local_dchi_dp over warp and atomic_add to shared dchi_dp for galaxy
	    	
*/

/*
class Patch {
    // Exposure data[], ierr[], xpix[], ypix[], astrometry
    // List of FixedGalaxy
    // Number of SkyGalaxy
	int nActiveGals; 
	int nFixedGals;
	SkyGalaxy * FixedGals;
	//..list of fixed galaxies? 
	PixFloat * data;
	PixFloat * ierr;
	PixFloat * xpix; 
	PixFloat * ypix; 
	//..astrometry? 
	PSFGaussian * psfgauss;
	int * n_psf_gauss;   // Indexed by exposure
	int * start_psf_gauss;   // Indexed by exposure
};

class SkyGalaxy {
    // On-sky galaxy parameters
	// flux: total flux
	// ra: right ascension (degrees)
	// dec: declination (degrees)
	// q, pa: axis ratio squared and position angle
	// n: sersic index
	// r: half-light radius (arcsec)
	float flux; 
	float ra; 
	float dec;
	float q; 
	float pa; 
	float n;
	float r; 
};
class Proposal {
	SkyGalaxy * ActiveGals;     // List of SkyGalaxy, input for this HMC likelihood
};

typedef struct{
	int nGauss; 
	ImageGaussian * gaussians; //List of ImageGaussians
    // TODO: May not need this to be explicit
} Galaxy;
*/  
//=================== ABOVE THIS LINE IS DEPRECATED ============


#include "header.hh"
#include "patch.cu"
#include "proposal.cu"

//NAM do we want this class, or should we make the convolve a method of PSFSourceGaussian?
typedef struct { //NAM TODO write a constructor? 
    // 6 Gaussian parameters
	float amp;
	float xcen;
	float ycen;
%%% 	float Cxx;
%%% 	float Cyy;
%%% 	float Cxy;
	
	float flux; 
	float G; 
	matrix22 CW; 
	
	matrix22 T; 
	matrix22 dT_dq;
	matrix22 dT_dpa;
	
	matrix22 scovar_im; 
	matrix22 scovar;    // Is this just the rad2 value?
	
	float da_dn;
	float da_dr; 
} PixGaussian;


class ImageGaussian {
  public:
    // 6 Gaussian parameters
	float amp;
	float xcen; 
	float ycen;
	float fxx; 
	float fyy;
	float fxy; 
	
    // 15 Jacobian elements (Image -> Sky)
    float dA_dFlux;
    float dx_dAlpha;
    float dy_dAlpha;
    float dx_dDelta;
    float dy_dDelta;
    float dA_dQ;
    float dFxx_dQ;
    float dFyy_dQ;
    float dFxy_dQ;
    float dA_dPA;
    float dFxx_dPA;
    float dFyy_dPA;
    float dFxy_dPA;
    float dA_dSersic;
    float dA_drh;
};


// TODO: This needs to be updated for the Patch class
__device__ PixFloat ComputeResidualImage(float xp, float yp, PixFloat data, Patch patch, Galaxy galaxy); //NAM do we need patch, galaxy? 
{
	PixFloat residual = data;
	
	// TODO: Need to loop over Active and Fixed galaxies
	//loop over all image gaussians g. 
	for (int i = 0; i < galaxy.nGauss; i ++){ //NAM TODO nGauss may be derived from Patch class properties. 
		ImageGaussian g = galaxy.Gaussians[i]
		float dx = xp - g.xcen; 
		float dy = yp - g.ycen; 
		float vx = g.fxx * dx + g.fxy * dy;
		float vy = g.fyy * dy + g.fxy * dx;
		float exparg = dx*vx+dy*vy;
		if (exparg>MAX_EXP_ARG) continue;
		float Gp = exp(-0.5 * exparg);

		// Here are the second-order corrections to the pixel integral
		float H = 1.0 + (vx*vx + vy*vy - g.fxx - g.fyy) / 24.0; 
		float C = g.amp * Gp * H; //count in this pixel. 
		
		residual -= C; 
	}
	return residual;
}

__device__ void ComputeGaussianDerivative(float xp, float yp, float residual_ierr2, 
            ImageGaussian *gauss, float * dchi2_dp) 
{
	float dx = xp - gauss->xcen; 
	float dy = yp - gauss->ycen; 
	float vx = gauss->fxx * dx + gauss->fxy * dy;
	float vy = gauss->fyy * dy + gauss->fxy * dx;
	float Gp = exp(-0.5 * (dx*vx + dy*vy));
	float H = 1.0; 
	
	float H = 1.0 + (vx*vx + vy*vy - gauss->fxx - gauss->fyy) / 24.0; 
	float C = residual_ierr2 * gauss->amp * Gp * H;   //count in this pixel. 
	
    float dC_dA = C / gauss->amp;
    float dC_dx = C*vx;
    float dC_dy = C*vy;
    float dC_dfx = -0.5*C*dx*dx;
    float dC_dfy = -0.5*C*dy*dy;
    float dC_dfxy = -1.0*C*dx*dy;
	
    float c_h = C / H;
    dC_dx -= c_h * (gauss->fxx*vx + gauss->fxy*vy) / 12.0;
    dC_dy -= c_h * (gauss->fyy*vy + gauss->fxy*vx) / 12.0;
    dC_dfx -= c_h * (1.0 - 2.0*dx*vx) / 24.0;
    dC_dfy -= c_h * (1.0 - 2.0*dy*vy) / 24.0;
    dC_dfxy += c_h * (dy*vx + dx*vy) / 12.0;
			 
    // TODO: Multiply by Jacobian and add to dchi2_dp
}


class Accumulator {
  public:
    float chi2;
    float dchi2_dp[NPARAM*MAXACTIVE]; //TODO: Need to figure out how to make this not compile time.

    Accumulator() {
        chi2 = 0.0;
        for (int j=0; j<NPARAM*MAXACTIVE; j++) dchi2_dp[j] = 0.0;
    }
    ~Accumulator() { }

    void warpReduceSum(float *answer, float input) {
        input += __shfl_down(input, 16);
        input += __shfl_down(input,  8);
        input += __shfl_down(input,  4);
        input += __shfl_down(input,  2);
        input += __shfl_down(input,  1);
        if (threadIdx.x&31==0) atomicAdd(answer, input);
    }
    
    // Could put the Reduction code in here
    void SumChi2(float _chi2) { warpReduceSum(&chi2, _chi2); }
    void SumDChi2dp(float *_dchi2_dp, int gal) { 
        for (int j=0; j<NPARAM; j++) 
            warpReduceSum(dchi2_dp+NPARAM*gal+j, _dchi2_dp[j]); 
    }

    /// This copies this Accumulator into another memory buffer
    inline void store(float *pchi2, float *pdchi2_dp, int nActive) {
        if (threadIdx.x==0) *pchi2 = chi2;
        for (int j=threadIdx.x; j<nActive*NPARAM; j+=BlockDim.x)
            pdchi2_dp[j] = dchi2_dp[j];
    }

    inline void addto(Accumulator &A) {
        if (threadIdx.x==0) chi2 += A.chi2;
        for (int j=threadIdx.x; j<nActive*NPARAM; j+=BlockDim.x)
            dchi2_dp[j] += A.dchi2_dp[j];
    }

    void coadd_and_sync(Accumulator *A, int nAcc) {
        for (int n=1; n<nAcc; n++) addto(A[n]);
        __syncthreads();
    }
};


__device__ void  GetGaussianAndJacobian(PixGaussian sersicgauss, PSFSourceGaussian psfgauss, ImageGaussian & gauss){
	sersicgauss.scovar_im = T * sersicgauss.scovar * T.T();   // TODO: Op provided
	
	matrix22 covar = sersicgauss.scovar_im + matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
	matrix22 f = covar.inv(); 
	
	gauss.fxx = f.v11; 
	gauss.fxy = f.v21; 
	gauss.fyy = f.v22; 
	
	gauss.xcen = sersicgauss.xcen + psfgauss.xcen; 
	gauss.ycen = sersicgauss.ycen + psfgauss.ycen; 
	
	gauss.amp = sersicgauss.flux * sersicgauss.amp * psfgauss.amp * sqrt(f.det()) / (2.0 * math.pi) ;
	
    // TODO: Check if this is duplicated
%%% 	matrix22 Sigma = sersicgauss.scovar_im + matrix22(psfgauss.Cxx, psfgauss.Cxy, psfgauss.Cxy, psfgauss.Cyy); 
%%% 	matrix22 F = Sigma.inv(); 
%%% 	float detF = F.det(); 
%%% 	float K = sersicgauss.flux * sersicgauss.G * sersicgauss.amp * psfgauss.amp * pow(detF, 0.5) / (2.0 * math.pi); 
	
	//now get derivatives 
	//of F
	matrix22 dSigma_dq  = sersicgauss.T * sersicgauss.scovar * sersicgauss.dT_dq.T() + sersicgauss.dT_dq * sersicgauss.scovar * sersicgauss.T.T(); 
	matrix22 dSigma_dpa = sersicgauss.T * sersicgauss.scovar * sersicgauss.dT_dpa.T() + sersicgauss.dT_dpa * sersicgauss.scovar * sersicgauss.T.T(); 
	matrix22 dF_dq      = -F * dSigma_dq * F; 
	matrix22 dF_dpa     = -F * dSigma_dpa * F; 
	float ddetF_dq   = detF *  (Sigma * dF_dq).trace(); 
	float ddetF_dpa  = detF * (Sigma * dF_dpa).trace(); 
	
	//of Amplitude
    gauss.dA_dQ = K / (2.0 * detF) * ddetF_dq;  
    gauss.dA_dpA = K / (2.0 * detF) * ddetF_dpa;  
    gauss.dA_dFlux = K / sersicgauss.flux; 
    gauss.dA_dSersic = K / sersicgauss.amp * sersicgauss.da_dn;
    gauss.dA_drh = K / sersicgauss.amp * sersicgauss.da_dr;
	
	gauss.dx_dAlpha = CW.v11; 
	gauss.dy_dAlpha = CW.v21; 
	
	gauss.dx_dDelta = CW.v12;
	gauss.dy_dDelta = CW.v22; 
	
	gauss.dFxx_dQ = dF_dq.v11;
	gauss.dFyy_dQ = dF_dq.v22;
	gauss.dFxy_dQ = dF_dq.v21; 

	gauss.dFxx_dPA = dF_dpa.v11;
	gauss.dFyy_dPA = dF_dpa.v22;
	gauss.dFxy_dPA = dF_dpa.v21; 
}



// __device__ void ConstructImageGaussian(matrix22 scovar, matrix22 pcovar,
// 												matrix22 smean, float samp, matrix22 pmean, float pamp, float flux, ImageGaussian &gauss){
// 	matrix22 covar = scovar + pcovar;
// 	matrix22 f = covar.inv();
//
// 	gauss.fxx = f.v11;
// 	gauss.fxy = f.v21;
// 	gauss.fyy = f.v22;
//
// 	gauss.xcen = smean.v11 + pmean.v11; //NAM careful!
// 	gauss.ycen = smean.v22 + pmean.v22;
//
// 	gauss.amp = flux * samp * pamp * pow(f.det(), 0.5) / (2.0 * math.pi) ;
// }
//
// __device__ void ConstructImageJacobian(matrix22 scovar, matrix22 pcovar,
// 												float samp, float pamp, float flux, float G, matrix22 T, matrix22 dT_dq, matrix22 dT_dpa, float da_dn, float da_dr, matrix22 CW, ImageGaussianJacobian &jacobian){
//
// 	//convolve the s-th Source component with the p-th PSF component.
// 	matrix22 scovar_im = T * scovar * T.T();
// 	matrix22 Sigma = scovar_im + pcovar;
// 	matrix22 F = Sigma.inv();
// 	float detF = F.det();
// 	float K = flux * G * samp * pamp * pow(detF, 0.5) / (2.0 * math.pi);
//
// 	//now get derivatives
// 	//of F
// 	matrix22 dSigma_dq  = T * scovar * dT_dq.T() + dT_dq * scovar * T.T();
// 	matrix22 dSigma_dpa = T * scovar * dT_dpa.T() + dT_dpa * scovar * T.T();
// 	matrix22 dF_dq      = -F * dSigma_dq * F;
// 	matrix22 dF_dpa     = -F * dSigma_dpa * F;
// 	float ddetF_dq   = detF *  (Sigma * dF_dq).trace();
// 	float ddetF_dpa  = detF * (Sigma * dF_dpa).trace();
//
// 	//of Amplitude
//     jacobian.dA_dQ = K / (2.0 * detF) * ddetF_dq;
//     jacobian.dA_dpA = K / (2.0 * detF) * ddetF_dpa;
//     jacobian.dA_dFlux = K / flux;
//     jacobian.dA_dSersic = K / samp * da_dn;
//     jacobian.dA_drh = K / samp * da_dr;
//
// 	jacobian.dx_dAlpha = CW.v11;
// 	jacobian.dy_dAlpha = CW.v21;
//
// 	jacobian.dx_dDelta = CW.v12;
// 	jacobian.dy_dDelta = CW.v22;
//
// 	jacobian.dFxx_dQ = dF_dq.v11;
// 	jacobian.dFyy_dQ = dF_dq.v22;
// 	jacobian.dFxy_dQ = dF_dq.v21;
//
// 	jacobian.dFxx_dPA = dF_dpa.v11;
// 	jacobian.dFyy_dPA = dF_dpa.v22;
// 	jacobian.dFxy_dPA = dF_dpa.v21;
// }

__device__ void CreateImageGaussians(Patch * patch, Source * sources, int exposure) {
	
    // TODO: These initializers may need to be only run in one thread, and then sync'ed
    __shared__ int band = blockIdx.x;   // This block is doing one band
	__shared__ int psfgauss_start = patch->psfgauss_start[exposure];
	__shared__ float G = patch->G[exposure]; 
	__shared__ float crpix[2], crval[2]; 
	
	crpix[0] = patch->crpix[exposure][0];  crpix[1] = patch->crpix[exposure][1];  
	crval[0] = patch->crval[exposure][0];  crval[1] = patch->crval[exposure][1]; 
	
	__shared__ int n_psf_per_source = patch->n_psf_per_source[band]; //constant per band. 
	__shared__ int n_radii = patch->n_radii;
    __shared__ int n_gal_gauss = patch->n_sources * n_psf_per_source;
    // TODO: Consider use of __constant__ variables

	for (int tid = threadIdx.x; tid < n_gal_gauss; tid += blockDim.x) {
        int g = tid / n_psf_per_source;  // Source number
		int p = tid - g * n_psf_per_source;   // Gaussian number
		
        // TODO: Convert . to ->
		Source *galaxy = sources+g; 	
		PSFSourceGaussian *psfgauss = patch->psfgauss+psfgauss_start + p; 
		PixGaussian	sersicgauss; 
		
		sersicgauss.G = G; 
		
		int s = psfgauss.sersic_radius_bin; 
			
	    // Do the setup of the transformations		
		//Get the transformation matrix and other conversions
		matrix22 D, R, S; 
		
		int d_cw_start = 4 * (patch->n_sources * exposure + g); 
		D  = matrix22(patch->D+d_cw_start);
		sersicgauss.CW = matrix22(patch->CW+d_cw_start);
		
		R.rot(galaxy.pa); 
		S.scale(galaxy.q); 
		sersicgauss.T = D * R * S; 
	
		//And its derivatives with respect to scene parameters
		matrix22 dS_dq, dR_dpa;
		dS_dq.scale_matrix_deriv(galaxy.q);
		dR_dpa.rotation_matrix_deriv(galaxy.pa);
		sersicgauss.dT_dq  = D * R * dS_dq; 
		sersicgauss.dT_dpa = D * dR_dpa * S; 	
	
		//NAM  might benefit from a vector class. this is gross. 
		float smean[2]; 
		smean[0] = galaxy.ra  - crval[0];
		smean[1] = galaxy.dec - crval[1]; 
	    matrix22::Av(CW, *smean);
		
		sersicgauss.xcen = smean[0] + crpix[0]; 
		sersicgauss.ycen = smean[1] + crpix[1]; 
		
        // TODO: Delete Cxx, Cyy, Cxy in PixGaussian
		sersicgauss.scovar = matrix22(patch->rad2[s], patch->rad2[s]);
		sersicgauss.amp = galaxy.mixture_amplitudes[s]; 
		
		sersicgauss.da_dn = galaxy.damplitude_dnsersic[s];
		sersicgauss.da_dr = galaxy.damplitude_drh[s] ; 

		//pull the correct flux from the multiband array
		sersicgauss.flux = galaxy->fluxes[band];


    	GetGaussianAndJacobian(sersicgauss, psfgauss, imageGauss[gal * n_psf_per_source + p]);
				
    	//ConstructImageGaussian(T * scovar T.T(), pcovar, smean, samp, pmean, pamp, flux, imageGauss[gal*n_gal_gauss+s*n_psf_gauss+p]);
        //ConstructImageJacobian(scovar, pcovar, samp, pamp, flux, G, T, dT_dq, dT_dpa, da_dn, da_dr, CW, imageJacob[gal*n_gal_gauss+s*n_psf_gauss+p]);
	}
}
	
	


// ================= Primary Proposal Kernel ========================

// Shared memory is arranged in 32 banks of 4 byte stagger

/// We are being handed pointers to a Patch structure, a Proposal structure,

/// a scalar chi2 response, and a vector dchi2_dp response.
/// The proposal is a pointer to Source[n_active] sources.
/// The response is a pointer to [band][MaxSource] responses.

__global__ void EvaluateProposal(void *_patch, void *_proposal, 
                                 void *pchi2, void *pdchi2_dp) {
    // Get the patch set up
    Patch *patch = (Patch *)_patch;  

    // The Proposal is a vector of Sources[n_active]
    Source *sources = (Source *)_proposal;

    // TODO: THIS IS BROKEN.
    // Need to define a shared pointer and then have one thread
    // call malloc to allocate this shared memory.
    // CreateAndZeroAccumulators();
    __shared__ Accumulator accum[NUMACCUMS]();
    int warp = threadIdx.x / ACCUMSIZE;  // We are accumulating each warp separately. 
	
    int band = blockIdx.x;   // This block is doing one band

    // Loop over Exposures
    for (int e = 0; e < patch->band_N[band]; e++) {
        int exposure = patch->band_start[band] + e;
		int start_psf_gauss = patch->psfgauss_start[exposure];

        // TODO: THIS IS BROKEN.
        // Need to define a shared pointer and then have one thread
        // call malloc to allocate this shared memory.
		int n_gal_gauss = patch->n_psf_per_source[band];
		__shared__ ImageGaussian imageGauss[n_gal_gauss * patch->n_sources];
            // [source][gauss]

        CreateImageGaussians(patch, sources, exposure, imageGauss);

		__syncthreads();
	
		for (int p = threadIdx.x ; p < patch->exposure_N[exposure]; p += blockDim.x) {
		    int pix = patch->exposure_start[exposure] + p;

		    float xp = patch->xpix[pix];
		    float yp = patch->ypix[pix];
		    PixFloat data = patch->data[pix];
		    PixFloat ierr = patch->ierr[pix];
		    PixFloat residual = ComputeResidualImage(xp, yp, data, imageGauss); 
            patch->residual[pix] = residual;
		    // This loads data and ierr, then subtracts the active
		    // and fixed Gaussians to make the residual

            residual *= ierr;   // Form residual/sigma, which is chi
		    float chi2 = residual*residual;
		    accum[warp].SumChi2(chi2);
		    /// ReduceWarp_Add(chi2, accum[warp].chi2));
            residual *= ierr;   // We want res*ierr^2 for the derivatives
	    
		    // Now we loop over Active Galaxies and compute the derivatives
		    for (int gal = 0; gal < patch.n_sources; gal++) {
                float dchi2_dp[NPARAM];
				for (int j=0; j<NPARAM; j++) dchi2_dp[j]=0.0;
				for (int gauss = 0; gauss<n_gal_gauss; gauss++) {  
				    ComputeGaussianDerivative(xp, yp, residual, imageGauss+gal*n_gal_gauss+gauss, dchi2_dp); 
				}
				accum[warp].SumDChi2dp(dchi2_dp, gal);
		    }
		}
	__syncthreads();
    }

    // Now we're done with all exposures, but we need to sum the Accumulators
    // over all warps.
    accum[0].coadd_and_sync(accum, blockDim.x/ACCUMSIZE);
    Response *r = (Response *)pdchi2_dp;
    accum[0].store((float *)pchi2, &(pdchi2_dp[blockIdx.x].dchi2_dparam), patch->n_sources);
    return;
}
