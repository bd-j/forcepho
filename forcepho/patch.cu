#include "hip/hip_runtime.h"
/* patch.cu

This is the data model for the Patch class on the GPU.  A Patch contains
all exposures for all bands in a patch.  It also contains per-source
astrometric information about how to convert from on-sky gaussians
to image gaussians.

PyCUDA will build this struct on the GPU side from data in the Python
Patch class (patch.py) and pass it as a pointer to each likelihood kernel
call.

TODO: decide how to fill this struct.  CUDA memcpy, or constructor kernel?

*/

#include <cstdint>

typedef float PixFloat;  // maybe put this elsewhere?

class Patch {
public:

    /* Image info */
    
    // Pixel data -- all exposures, all bands
    // Note that the pixels are organized into warp-sized compact superpixels
    // and exposures are padded to blockDim.x.
    PixFloat *data;
    PixFloat *ierr;
    PixFloat *xpix;
    PixFloat *ypix;
    PixFloat *residual;

    /* Indexing for the image data */
    // Number of bands and exposures is known from the CUDA grid size
    // int n_bands = gridDim.x

    // These index the pixel arrays
    int *exposure_start;    // [expnum]
    int *exposure_N;        // [expnum]  

    // These index the exposure_start and exposure_N arrays
    // bands are indexed sequentially, not by any band ID
    // These are the expnum used elsewhere
    int16_t *band_start;    // [band]
    int16_t *band_N;        // [band]

    // ------------------ Source data --------------------
    // Number of active sources
    // (GPU never knows about inactive sources)
    int n_sources;

    // The number of radii we're using in our Sersic models
    int n_radii;   

    // ----------------------- Astrometry --------------------
    // Astrometry: scale, rotation matrices (and derivatives)
    // See gaussmodel.py: convert_to_gaussians() for how these are used.
    // If D is a 2x2 matrix, then the index of matrix element (i,j) is:
    //      D[4*nsource*exposure + 4*source + 2*i + j]
    // (could also be an array of matrix structs if preferred)
    // The exposure indices for a band can be found from band_start and band_N
    
    // D is pixels per arcsec, d(pixel x,y)/d(sky).
    // Here the sky is in arcseconds of displacement, which differs from CW
    // because of a cos(dec)
    float *D;       // [expnum][source][2][2]

    // The Coordinate Reference Point has a pixel location and a RA/Dec
    float *crpix;   // [expnum][2] -- Image pixel
    float *crval;   // [expnum][2] -- RA/Dec 

    // CW is d(pixel x,y)/d(RA,dec) expanded around CR point
    float *CW;      // [expnum][source][2][2]

    // G is the conversion from our sky flux scale into exposure counts
    float *G;       // [expnum]  


    // --------------  PSF Gaussians  ---------------------

    // The number of PSFSourceGaussians per source per exposure
    // This number is constant for a given band, hence this array is length nbands
    int *n_psf_per_source;  [band] // NOTE: This could have been type int8_t

    // Few per sersic bin per source per exposure
    // Indexing is:  TODO: Fix below
    //      psfgauss[exposure*n_psf_per_source[band]*nsource + n_psf_per_source[band]*source + psf]
    // The exposure indices for a band can be found from band_start and band_N
    PSFSourceGaussian *psfgauss;   [expnum][source][psfgauss_per_source]
    int *psfgauss_start;    [expnum]
    // psfgauss_N = n_psf_per_source*n_sources
};






class PSFSourceGaussian {
    /*
    Describes a single Gaussian that has already been convolved
    with a source Gaussian of a certain sersic radius bin.
    */
public:

    // Gaussian parameters
    float amp;
    float xcen,ycen;
    float Cxx, Cyy, Cxy;

    // The index of the sersic radius bin this Gaussian applies to
    int sersic_radius_bin;
};
